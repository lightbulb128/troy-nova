#include "hip/hip_runtime.h"
#include "constants.h"
#include "rns_tool.h"
#include "uint_small_mod.h"
#include "polynomial_buffer.h"

#include "../fgk/rns_tool.h"

namespace troy {namespace utils {

    static void print_array(ConstSlice<uint64_t> array, bool end_line = true) {
        if (array.on_device()) {
            Array<uint64_t> host = Array<uint64_t>::create_and_copy_from_slice(array, false, nullptr);
            print_array(host.const_reference(), end_line);
            return;
        }
        std::cout << "[";
        for (size_t i = 0; i < array.size(); i++) {
            std::cout << array[i];
            if (i != array.size() - 1) {
                std::cout << ", ";
            }
        }
        std::cout << "]";
        if (end_line) {
            std::cout << std::endl;
        }
    }
    
    RNSTool::RNSTool(size_t poly_modulus_degree, const RNSBase& q, const Modulus& t) {

        if (q.on_device()) {
            throw std::invalid_argument("[RNSTool::RNSTool] Cannot create RNSTool from device RNSBase q.");
        }

        if (q.size() < utils::HE_COEFF_MOD_COUNT_MIN || q.size() > utils::HE_COEFF_MOD_COUNT_MAX) {
            throw std::invalid_argument("[RNSTool::RNSTool] RNSBase length is invalid.");
        }

        int coeff_count_power = utils::get_power_of_two(poly_modulus_degree);

        if (
            coeff_count_power < 0 || 
            poly_modulus_degree > utils::HE_POLY_MOD_DEGREE_MAX ||
            poly_modulus_degree < utils::HE_POLY_MOD_DEGREE_MIN
        ) {
            throw std::invalid_argument("[RNSTool::RNSTool] Polynomial modulus degree is invalid.");
        }
        
        // Allocate memory for the bases q, B, Bsk, Bsk U m_tilde, t_gamma
        size_t base_q_size = q.size();

        // In some cases we might need to increase the size of the base B by one, namely we require
        // K * n * t * q^2 < q * prod(B) * m_sk, where K takes into account cross terms when larger size ciphertexts
        // are used, and n is the "delta factor" for the ring. We reserve 32 bits for K * n. Here the coeff modulus
        // primes q_i are bounded to be HE_USER_MOD_BIT_COUNT_MAX (60) bits, and all primes in B and m_sk are
        // HE_INTERNAL_MOD_BIT_COUNT (61) bits.
        size_t total_coeff_bit_count = utils::get_significant_bit_count_uint(q.base_product());

        size_t base_B_size = base_q_size;
        if (32 + t.bit_count() + total_coeff_bit_count >= utils::HE_INTERNAL_MOD_BIT_COUNT * base_q_size + utils::HE_INTERNAL_MOD_BIT_COUNT) {
            base_B_size++;
        }

        size_t base_Bsk_size = base_B_size + 1;
        size_t base_Bsk_m_tilde_size = base_Bsk_size + 1;

        // Sample primes for B and two more primes: m_sk and gamma
        size_t coeff_count = poly_modulus_degree;
        
        std::vector<Modulus> baseconv_primes = utils::get_primes(
            static_cast<uint64_t>(2 * coeff_count),
            utils::HE_INTERNAL_MOD_BIT_COUNT,
            base_Bsk_m_tilde_size
        );
        Modulus m_sk = baseconv_primes[0];
        Modulus gamma = baseconv_primes[1];
        std::vector<Modulus> base_B_primes; base_B_primes.reserve(baseconv_primes.size() - 2);
        for (size_t i = 2; i < baseconv_primes.size(); i++) {
            base_B_primes.push_back(baseconv_primes[i]);
        }

        // Set m_tilde to a non-prime value
        Modulus m_tilde(1ul << 32);
        uint64_t m_tilde_value = m_tilde.value();
        
        // Populate the base arrays
        RNSBase base_q = q.clone(nullptr);
        RNSBase base_B(ConstSlice(base_B_primes.data(), base_B_primes.size(), false, nullptr));
        RNSBase base_Bsk = base_B.extend_modulus(m_sk);
        RNSBase base_Bsk_m_tilde = base_Bsk.extend_modulus(m_tilde);

        // Set up t-gamma base if t_ is non-zero (using BFV)
        std::optional<RNSBase> base_t_gamma = std::nullopt;
        std::optional<BaseConverter> base_q_to_t_conv = std::nullopt;
        if (!t.is_zero()) {
            Modulus t_gamma[2]{ t, gamma };
            base_t_gamma = std::optional(RNSBase(ConstSlice(t_gamma, 2, false, nullptr)));
            base_q_to_t_conv = std::optional(BaseConverter(base_q, RNSBase(ConstSlice(&t, 1, false, nullptr))));
        }
        
        // Generate the Bsk NTTTables; these are used for NTT after base extension to Bsk
        Array<NTTTables> base_Bsk_ntt_tables = NTTTables::create_ntt_tables(
            coeff_count_power,
            base_Bsk.base()
        );

        BaseConverter base_q_to_Bsk_conv = BaseConverter(base_q, base_Bsk);
        BaseConverter base_q_to_m_tilde_conv = BaseConverter(base_q, RNSBase(ConstSlice(&m_tilde, 1, false, nullptr)));
        BaseConverter base_B_to_q_conv = BaseConverter(base_B, base_q);
        BaseConverter base_B_to_m_sk_conv = BaseConverter(base_B, RNSBase(ConstSlice(&m_sk, 1, false, nullptr)));
        std::optional<BaseConverter> base_q_to_t_gamma_conv = std::nullopt;
        if (base_q_to_t_conv.has_value()) {
            base_q_to_t_gamma_conv = std::optional(BaseConverter(base_q, base_t_gamma.value()));
        }

        // Compute prod(B) mod q
        Array<uint64_t> prod_B_mod_q(base_q.size(), false, nullptr);
        for (size_t i = 0; i < base_q.size(); i++) {
            prod_B_mod_q[i] = utils::modulo_uint(base_B.base_product(), base_q.base()[i]);
        }

        // Compute prod(q)^(-1) mod Bsk
        Array<MultiplyUint64Operand> inv_prod_q_mod_Bsk(base_Bsk.size(), false, nullptr);
        for (size_t i = 0; i < base_Bsk.size(); i++) {
            const Modulus& modulus = base_Bsk.base()[i];
            uint64_t temp = utils::modulo_uint(base_q.base_product(), modulus);
            bool try_invert = utils::try_invert_uint64_mod(temp, modulus, temp);
            if (!try_invert) {
                throw std::invalid_argument("[RNSTool::RNSTool] Unable to invert base_q product.");
            }
            inv_prod_q_mod_Bsk[i] = MultiplyUint64Operand(temp, modulus);
        }

        // Compute prod(B)^(-1) mod m_sk
        uint64_t temp = utils::modulo_uint(base_B.base_product(), m_sk);
        bool try_invert = utils::try_invert_uint64_mod(temp, m_sk, temp);
        if (!try_invert) {
            throw std::invalid_argument("[RNSTool::RNSTool] Unable to invert base_B product.");
        }
        MultiplyUint64Operand inv_prod_B_mod_m_sk(temp, m_sk);

        // Compute m_tilde^(-1) mod Bsk
        Array<MultiplyUint64Operand> inv_m_tilde_mod_Bsk(base_Bsk.size(), false, nullptr);
        for (size_t i = 0; i < base_Bsk.size(); i++) {
            const Modulus& modulus = base_Bsk.base()[i];
            try_invert = utils::try_invert_uint64_mod(modulus.reduce(m_tilde.value()), modulus, temp);
            if (!try_invert) {
                throw std::invalid_argument("[RNSTool::RNSTool] Unable to invert m_tilde.");
            }
            inv_m_tilde_mod_Bsk[i] = MultiplyUint64Operand(temp, modulus);
        }
        
        // Compute prod(q)^(-1) mod m_tilde
        temp = utils::modulo_uint(base_q.base_product(), m_tilde);
        try_invert = utils::try_invert_uint64_mod(temp, m_tilde, temp);
        if (!try_invert) {
            throw std::invalid_argument("[RNSTool::RNSTool] Unable to invert base_q product.");
        }
        MultiplyUint64Operand neg_inv_prod_q_mod_m_tilde(
            utils::negate_uint64_mod(temp, m_tilde), m_tilde
        );

        Array<uint64_t> prod_q_mod_Bsk(base_Bsk.size(), false, nullptr);
        for (size_t i = 0; i < base_Bsk.size(); i++) {
            prod_q_mod_Bsk[i] = utils::modulo_uint(base_q.base_product(), base_Bsk.base()[i]);
        }

        std::optional<MultiplyUint64Operand> inv_gamma_mod_t = std::nullopt;
        std::optional<Array<MultiplyUint64Operand>> prod_t_gamma_mod_q = std::nullopt;
        std::optional<Array<MultiplyUint64Operand>> neg_inv_q_mod_t_gamma = std::nullopt;
        uint64_t inv_q_last_mod_t =1;
        uint64_t q_last_mod_t = 1;
        if (base_t_gamma.has_value()) {

            // Compute gamma^(-1) mod t
            try_invert = utils::try_invert_uint64_mod(t.reduce(gamma.value()), t, temp);
            if (!try_invert) {
                throw std::invalid_argument("[RNSTool::RNSTool] Unable to invert gamma mod t.");
            }
            inv_gamma_mod_t = std::optional(MultiplyUint64Operand(temp, t));
            
            // Compute prod({t, gamma}) mod q
            prod_t_gamma_mod_q = std::optional(Array<MultiplyUint64Operand>(base_q.size(), false, nullptr));
            for (size_t i = 0; i < base_q.size(); i++) {
                const Modulus& modulus = base_q.base()[i];
                prod_t_gamma_mod_q.value()[i] = MultiplyUint64Operand(
                    utils::multiply_uint64_mod(
                        base_t_gamma.value().base()[0].value(),
                        base_t_gamma.value().base()[1].value(),
                        modulus
                    ),
                    modulus
                );
            }

            // Compute -prod(q)^(-1) mod {t, gamma}
            neg_inv_q_mod_t_gamma = std::optional(Array<MultiplyUint64Operand>(2, false, nullptr));
            for (size_t i = 0; i < 2; i++) {
                const Modulus& modulus = base_t_gamma.value().base()[i];
                temp = utils::modulo_uint(base_q.base_product(), modulus);
                try_invert = utils::try_invert_uint64_mod(temp, modulus, temp);
                if (!try_invert) {
                    throw std::invalid_argument("[RNSTool::RNSTool] Unable to invert base_q mod t_gamma.");
                }
                neg_inv_q_mod_t_gamma.value()[i] = MultiplyUint64Operand(
                    utils::negate_uint64_mod(temp, modulus),
                    modulus
                );
            }
        }

        // Compute q[last]^(-1) mod q[i] for i = 0..last-1
        // This is used by modulus switching and rescaling
        Array<MultiplyUint64Operand> inv_q_last_mod_q(base_q.size() - 1, false, nullptr);
        const Modulus& last_q = base_q.base()[base_q.size() - 1];
        for (size_t i = 0; i < base_q.size() - 1; i++) {
            const Modulus& modulus = base_q.base()[i];
            try_invert = utils::try_invert_uint64_mod(last_q.value(), modulus, temp);
            if (!try_invert) {
                throw std::invalid_argument("[RNSTool::RNSTool] Unable to invert q[last] mod q[i].");
            }
            inv_q_last_mod_q[i] = MultiplyUint64Operand(temp, modulus);
        }

        if (!t.is_zero()) {
            try_invert = utils::try_invert_uint64_mod(last_q.value(), t, temp);
            if (!try_invert) {
                throw std::invalid_argument("[RNSTool::RNSTool] Unable to invert q[last] mod t.");
            }
            inv_q_last_mod_t = temp;
            q_last_mod_t = t.reduce(last_q.value());
        }

        // set the members

        this->coeff_count_ = coeff_count;

        this->base_q_ = std::move(base_q);
        this->base_B_ = std::move(base_B);
        this->base_Bsk_ = std::move(base_Bsk);
        this->base_Bsk_m_tilde_ = std::move(base_Bsk_m_tilde);
        this->base_t_gamma_ = std::move(base_t_gamma);

        this->base_q_to_Bsk_conv_ = std::move(base_q_to_Bsk_conv);
        this->base_q_to_m_tilde_conv_ = std::move(base_q_to_m_tilde_conv);
        this->base_B_to_q_conv_ = std::move(base_B_to_q_conv);
        this->base_B_to_m_sk_conv_ = std::move(base_B_to_m_sk_conv);
        this->base_q_to_t_gamma_conv_ = std::move(base_q_to_t_gamma_conv);
        this->base_q_to_t_conv_ = std::move(base_q_to_t_conv);

        this->inv_prod_q_mod_Bsk_ = std::move(inv_prod_q_mod_Bsk);
        this->neg_inv_prod_q_mod_m_tilde_ = std::move(neg_inv_prod_q_mod_m_tilde);
        this->inv_prod_B_mod_m_sk_ = std::move(inv_prod_B_mod_m_sk);
        this->inv_gamma_mod_t_ = std::move(inv_gamma_mod_t);
        this->prod_B_mod_q_ = std::move(prod_B_mod_q);
        this->inv_m_tilde_mod_Bsk_ = std::move(inv_m_tilde_mod_Bsk);
        this->prod_q_mod_Bsk_ = std::move(prod_q_mod_Bsk);
        this->neg_inv_q_mod_t_gamma_ = std::move(neg_inv_q_mod_t_gamma);
        this->prod_t_gamma_mod_q_ = std::move(prod_t_gamma_mod_q);
        this->inv_q_last_mod_q_ = std::move(inv_q_last_mod_q);
        this->base_Bsk_ntt_tables_ = std::move(base_Bsk_ntt_tables);

        this->m_tilde_ = Box(new Modulus(m_tilde), false, nullptr);
        this->m_sk_ = Box(new Modulus(m_sk), false, nullptr);
        this->t_ = Box(new Modulus(t), false, nullptr);
        this->gamma_ = Box(new Modulus(gamma), false, nullptr);

        this->m_tilde_value_ = m_tilde_value;
        this->inv_q_last_mod_t_ = inv_q_last_mod_t;
        this->q_last_mod_t_ = q_last_mod_t;
        this->q_last_half_ = last_q.value() >> 1;

        this->device = false;
        
    }

    template <typename T>
    static std::optional<T> optional_clone(const std::optional<T>& opt, MemoryPoolHandle pool) {
        if (opt.has_value()) {
            return std::optional<T>(opt.value().clone(pool));
        }
        return std::nullopt;
    }

    RNSTool RNSTool::clone(MemoryPoolHandle pool) const {
        RNSTool cloned;

        cloned.coeff_count_ = this->coeff_count_;

        cloned.base_q_ = this->base_q_.clone(pool);
        cloned.base_B_ = this->base_B_.clone(pool);
        cloned.base_Bsk_ = this->base_Bsk_.clone(pool);
        cloned.base_Bsk_m_tilde_ = this->base_Bsk_m_tilde_.clone(pool);
        cloned.base_t_gamma_ = optional_clone(this->base_t_gamma_, pool);

        cloned.base_q_to_Bsk_conv_ = this->base_q_to_Bsk_conv_.clone(pool);
        cloned.base_q_to_m_tilde_conv_ = this->base_q_to_m_tilde_conv_.clone(pool);
        cloned.base_B_to_q_conv_ = this->base_B_to_q_conv_.clone(pool);
        cloned.base_B_to_m_sk_conv_ = this->base_B_to_m_sk_conv_.clone(pool);
        cloned.base_q_to_t_gamma_conv_ = optional_clone(this->base_q_to_t_gamma_conv_, pool);
        cloned.base_q_to_t_conv_ = optional_clone(this->base_q_to_t_conv_, pool);

        cloned.inv_prod_q_mod_Bsk_ = this->inv_prod_q_mod_Bsk_.clone(pool);
        cloned.neg_inv_prod_q_mod_m_tilde_ = this->neg_inv_prod_q_mod_m_tilde_;
        cloned.inv_prod_B_mod_m_sk_ = this->inv_prod_B_mod_m_sk_;
        cloned.inv_gamma_mod_t_ = this->inv_gamma_mod_t_;
        cloned.prod_B_mod_q_ = this->prod_B_mod_q_.clone(pool);
        cloned.inv_m_tilde_mod_Bsk_ = this->inv_m_tilde_mod_Bsk_.clone(pool);
        cloned.prod_q_mod_Bsk_ = this->prod_q_mod_Bsk_.clone(pool);
        cloned.neg_inv_q_mod_t_gamma_ = optional_clone(this->neg_inv_q_mod_t_gamma_, pool);
        cloned.prod_t_gamma_mod_q_ = optional_clone(this->prod_t_gamma_mod_q_, pool);
        cloned.inv_q_last_mod_q_ = this->inv_q_last_mod_q_.clone(pool);
        cloned.base_Bsk_ntt_tables_ = this->base_Bsk_ntt_tables_.clone(pool);
        cloned.m_tilde_ = this->m_tilde_.clone(pool);
        cloned.m_sk_ = this->m_sk_.clone(pool);

        cloned.t_ = this->t_.clone(pool);
        cloned.gamma_ = this->gamma_.clone(pool);
        cloned.m_tilde_value_ = this->m_tilde_value_;
        cloned.inv_q_last_mod_t_ = this->inv_q_last_mod_t_;
        cloned.q_last_mod_t_ = this->q_last_mod_t_;
        cloned.q_last_half_ = this->q_last_half_;

        cloned.device = this->device;

        return cloned;
    }

    template <typename T>
    static void optional_to_device_inplace(std::optional<T>& opt, MemoryPoolHandle pool) {
        if (opt.has_value()) {
            opt.value().to_device_inplace(pool);
        }
    }

    void RNSTool::to_device_inplace(MemoryPoolHandle pool) {
        if (this->on_device()) {
            return;
        }
        
        this->base_q_.to_device_inplace(pool);
        this->base_B_.to_device_inplace(pool);
        this->base_Bsk_.to_device_inplace(pool);
        this->base_Bsk_m_tilde_.to_device_inplace(pool);
        optional_to_device_inplace(this->base_t_gamma_, pool);

        this->base_q_to_Bsk_conv_.to_device_inplace(pool);
        this->base_q_to_m_tilde_conv_.to_device_inplace(pool);
        this->base_B_to_q_conv_.to_device_inplace(pool);
        this->base_B_to_m_sk_conv_.to_device_inplace(pool);
        optional_to_device_inplace(this->base_q_to_t_gamma_conv_, pool);
        optional_to_device_inplace(this->base_q_to_t_conv_, pool);

        this->inv_prod_q_mod_Bsk_.to_device_inplace(pool);
        this->prod_B_mod_q_.to_device_inplace(pool);
        this->inv_m_tilde_mod_Bsk_.to_device_inplace(pool);
        this->prod_q_mod_Bsk_.to_device_inplace(pool);
        optional_to_device_inplace(this->neg_inv_q_mod_t_gamma_, pool);
        optional_to_device_inplace(this->prod_t_gamma_mod_q_, pool);
        this->inv_q_last_mod_q_.to_device_inplace(pool);
        for (size_t i = 0; i < base_Bsk_ntt_tables_.size(); i++) {
            this->base_Bsk_ntt_tables_[i].to_device_inplace(pool);
        }
        this->base_Bsk_ntt_tables_.to_device_inplace(pool);

        this->m_tilde_.to_device_inplace(pool);
        this->m_sk_.to_device_inplace(pool);
        this->t_.to_device_inplace(pool);
        this->gamma_.to_device_inplace(pool);

        this->device = true;
    }

    __device__ static void device_divide_and_round_q_last(
        ConstSlice<Modulus> base_q, size_t coeff_count, size_t q_last_half,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q, 
        ConstSlice<uint64_t> input, size_t input_pcount, Slice<uint64_t> destination
    ) {
        size_t j = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_q_size = base_q.size();
        if (j >= coeff_count) return;

        for (size_t p = 0; p < input_pcount; p++) {
            size_t poffset = p * coeff_count * base_q_size;
            size_t doffset = p * coeff_count * (base_q_size - 1);
            uint64_t input_last = input[poffset + (base_q_size - 1) * coeff_count + j];
            uint64_t input_last_translated = utils::add_uint64_mod(input_last, q_last_half, base_q[base_q_size - 1]);
            for (size_t i = 0; i < base_q_size - 1; i++) {
                const Modulus& modulus = *base_q.at(i);
                // (ct mod qk) mod qi
                uint64_t temp = modulus.reduce(input_last_translated);
                // Subtract rounding correction here; the negative sign will turn into a plus in the next subtraction
                uint64_t half_mod = modulus.reduce(q_last_half);
                temp = utils::sub_uint64_mod(temp, half_mod, modulus);
                // (ct mod qi) - (ct mod qk) mod qi
                uint64_t input_ij = utils::sub_uint64_mod(input[poffset + i * coeff_count + j], temp, modulus);
                // qk^(-1) * ((ct mod qi) - (ct mod qk)) mod qi
                destination[doffset + i * coeff_count + j] = utils::multiply_uint64operand_mod(input_ij, inv_q_last_mod_q[i], modulus);
            }
        }

    }

    __global__ static void kernel_divide_and_round_q_last(
        ConstSlice<Modulus> base_q, size_t coeff_count, size_t q_last_half,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q, 
        ConstSlice<uint64_t> input, size_t input_pcount, Slice<uint64_t> destination
    ) {
        device_divide_and_round_q_last(base_q, coeff_count, q_last_half, inv_q_last_mod_q, input, input_pcount, destination);
    }

    __global__ static void kernel_divide_and_round_q_last_batched(
        ConstSlice<Modulus> base_q, size_t coeff_count, size_t q_last_half,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q, 
        ConstSliceArrayRef<uint64_t> input, size_t input_pcount, SliceArrayRef<uint64_t> destination
    ) {
        size_t i = blockIdx.y;
        device_divide_and_round_q_last(base_q, coeff_count, q_last_half, inv_q_last_mod_q, input[i], input_pcount, destination[i]);
    }

    void RNSTool::divide_and_round_q_last(ConstSlice<uint64_t> input, size_t input_pcount, Slice<uint64_t> destination) const {
        bool device = this->on_device();
        if (!utils::device_compatible(*this, input)) {
            throw std::invalid_argument("[RNSTool::divide_and_round_q_last_inplace] RNSTool and input must be on the same device.");
        }
        size_t base_q_size = this->base_q().size();
        size_t coeff_count = this->coeff_count();
        if (device) {
            size_t block_count = utils::ceil_div(coeff_count, utils::KERNEL_THREAD_COUNT);
            utils::set_device(input.device_index());
            kernel_divide_and_round_q_last<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                this->base_q().base(),
                this->coeff_count(),
                this->q_last_half(),
                this->inv_q_last_mod_q(),
                input,
                input_pcount, destination
            );
            utils::stream_sync();
        } else {
            size_t half = this->q_last_half();
            for (size_t p = 0; p < input_pcount; p++) {
                size_t poffset = p * coeff_count * base_q_size;
                size_t doffset = p * coeff_count * (base_q_size - 1);
                ConstSlice<uint64_t> input_last = input.const_slice(poffset + (base_q_size - 1) * coeff_count, poffset + base_q_size * coeff_count);
                Buffer<uint64_t> input_last_translated(coeff_count, false, nullptr);
                utils::add_scalar(input_last, this->q_last_half(), this->base_q().base().at(base_q_size - 1), input_last_translated.reference());
                Buffer<uint64_t> temp(coeff_count, false, nullptr);
                for (size_t i = 0; i < base_q_size - 1; i++) {
                    ConstPointer<Modulus> modulus = this->base_q().base().at(i);
                    ConstSlice<uint64_t> input_i = input.const_slice(poffset + i * coeff_count, poffset + (i + 1) * coeff_count);
                    Slice<uint64_t> dest_i = destination.slice(doffset + i * coeff_count, doffset + (i + 1) * coeff_count);
                    // (ct mod qk) mod qi
                    utils::modulo(input_last_translated.const_reference(), modulus, temp.reference());
                    // Subtract rounding correction here; the negative sign will turn into a plus in the next subtraction
                    uint64_t half_mod = modulus->reduce(half);
                    utils::sub_scalar_inplace(temp.reference(), half_mod, modulus);
                    // (ct mod qi) - (ct mod qk) mod qi
                    utils::sub(input_i, temp.const_reference(), modulus, dest_i);
                    // qk^(-1) * ((ct mod qi) - (ct mod qk)) mod qi
                    utils::multiply_uint64operand_inplace(dest_i, this->inv_q_last_mod_q().at(i), modulus);
                }
            }

        }
    }
    
    void RNSTool::divide_and_round_q_last_batched(const ConstSliceVec<uint64_t>& input, size_t input_pcount, const SliceVec<uint64_t>& destination, MemoryPoolHandle pool) const {
        if (input.size() != destination.size()) {
            throw std::invalid_argument("[RNSTool::divide_and_round_q_last_batched] input and destination must have the same number of elements.");
        }
        if (input.size() == 0) return;
        bool device = this->on_device();
        size_t n = input.size();
        if (!device || n < BATCH_OP_THRESHOLD) {
            for (size_t i = 0; i < n; i++) {
                this->divide_and_round_q_last(input[i], input_pcount, destination[i]);
            }
        } else {
            size_t coeff_count = this->coeff_count();
            size_t block_count = utils::ceil_div(coeff_count, utils::KERNEL_THREAD_COUNT);
            auto comp_ref = this->base_q().base();
            auto input_batched = construct_batch(input, pool, comp_ref);
            auto destination_batched = construct_batch(destination, pool, comp_ref);
            utils::set_device(comp_ref.device_index());
            dim3 block_dims(block_count, n);
            kernel_divide_and_round_q_last_batched<<<block_dims, utils::KERNEL_THREAD_COUNT>>>(
                this->base_q().base(),
                this->coeff_count(),
                this->q_last_half(),
                this->inv_q_last_mod_q(),
                input_batched,
                input_pcount, destination_batched
            );
            utils::stream_sync();
        }
    }

    static void host_divide_and_round_q_last_ntt_step1(const RNSTool& self, Slice<uint64_t> input, size_t pcount, Slice<uint64_t> temp) {
        size_t base_q_size = self.base_q().size();
        size_t coeff_count = self.coeff_count();
        for (size_t p = 0; p < pcount; p++) {
            size_t poffset = p * coeff_count * base_q_size;
            size_t toffset = p * coeff_count * (base_q_size - 1);
            Slice<uint64_t> input_last = input.slice(poffset + (base_q_size - 1) * coeff_count, poffset + base_q_size * coeff_count);
            utils::add_scalar_inplace(input_last, self.q_last_half(), self.base_q().base().at(base_q_size - 1));
            ConstPointer<Modulus> last_modulus = self.base_q().base().at(base_q_size - 1);

            for (size_t i = 0; i < base_q_size - 1; i++) {
                ConstPointer<Modulus> modulus = self.base_q().base().at(i);
                Slice<uint64_t> temp_i = temp.slice(toffset + i * coeff_count, toffset + (i + 1) * coeff_count);
                if (modulus->value() < last_modulus->value()) {
                    utils::modulo(input_last.as_const(), modulus, temp_i);
                } else {
                    utils::set_uint(input_last.as_const(), coeff_count, temp_i);
                }
                uint64_t half_mod = modulus->reduce(self.q_last_half());
                utils::sub_scalar_inplace(temp_i, half_mod, modulus);
            }
        }
    }

    __device__ static void device_divide_and_round_q_last_ntt_step1(
        ConstSlice<Modulus> base_q, size_t coeff_count, size_t q_last_half,
        ConstSlice<uint64_t> input, size_t pcount, Slice<uint64_t> temp
    ) {
        size_t j = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_q_size = base_q.size();
        if (j >= coeff_count) return;

        for (size_t p = 0; p < pcount; p++) {
            size_t poffset = p * coeff_count * base_q_size;
            size_t toffset = p * coeff_count * (base_q_size - 1);
            uint64_t input_last = input[poffset + (base_q_size - 1) * coeff_count + j];
            input_last = utils::add_uint64_mod(input_last, q_last_half, base_q[base_q_size - 1]);
            for (size_t i = 0; i < base_q_size - 1; i++) {
                const Modulus& last_modulus = *base_q.at(base_q_size - 1);
                uint64_t temp_value;
                const Modulus& modulus = *base_q.at(i);
                if (modulus.value() < last_modulus.value()) {
                    temp_value = modulus.reduce(input_last);
                } else {
                    temp_value = input_last;
                }
                uint64_t half_mod = modulus.reduce(q_last_half);
                temp_value = utils::sub_uint64_mod(temp_value, half_mod, modulus);
                temp[toffset + i * coeff_count + j] = temp_value;
            }
        }
    }

    __global__ static void kernel_divide_and_round_q_last_ntt_step1(
        ConstSlice<Modulus> base_q, size_t coeff_count, size_t q_last_half,
        ConstSlice<uint64_t> input, size_t pcount, Slice<uint64_t> temp
    ) {
        device_divide_and_round_q_last_ntt_step1(base_q, coeff_count, q_last_half, input, pcount, temp);
    }

    __global__ static void kernel_divide_and_round_q_last_ntt_step1_batched(
        ConstSlice<Modulus> base_q, size_t coeff_count, size_t q_last_half,
        ConstSliceArrayRef<uint64_t> input, size_t pcount, SliceArrayRef<uint64_t> temp
    ) {
        size_t i = blockIdx.y;
        device_divide_and_round_q_last_ntt_step1(base_q, coeff_count, q_last_half, input[i], pcount, temp[i]);
    }

    static void divide_and_round_q_last_ntt_step1(const RNSTool& self, Slice<uint64_t> input, size_t pcount, Slice<uint64_t> temp) {
        bool device = self.on_device();
        if (device) {
            size_t block_count = utils::ceil_div(self.coeff_count(), utils::KERNEL_THREAD_COUNT);
            utils::set_device(input.device_index());
            kernel_divide_and_round_q_last_ntt_step1<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                self.base_q().base(),
                self.coeff_count(),
                self.q_last_half(),
                input,
                pcount,
                temp
            );
            utils::stream_sync();
        } else {
            host_divide_and_round_q_last_ntt_step1(self, input, pcount, temp);
        }
    }
    
    static void host_divide_and_round_q_last_ntt_step2(const RNSTool& self, ConstSlice<uint64_t> input, Slice<uint64_t> destination, size_t pcount, ConstSlice<uint64_t> temp) {
        
        size_t base_q_size = self.base_q().size();
        size_t coeff_count = self.coeff_count();
        
        for (size_t p = 0; p < pcount; p++) {
            size_t poffset = p * coeff_count * base_q_size;
            size_t offset = p * coeff_count * (base_q_size - 1);
            for (size_t i = 0; i < base_q_size - 1; i++) {
                ConstPointer<Modulus> modulus = self.base_q().base().at(i);
                ConstSlice<uint64_t> input_i = input.const_slice(poffset + i * coeff_count, poffset + (i + 1) * coeff_count);
                Slice<uint64_t> dest_i = destination.slice(offset + i * coeff_count, offset + (i + 1) * coeff_count);
                ConstSlice<uint64_t> temp_i = temp.const_slice(offset + i * coeff_count, offset + (i + 1) * coeff_count);
                uint64_t qi_lazy = modulus->value() << 2;
                utils::add_scalar(input_i, qi_lazy, modulus, dest_i);
                utils::sub_inplace(dest_i, temp_i, modulus);
                utils::multiply_uint64operand_inplace(dest_i, self.inv_q_last_mod_q().at(i), modulus);
            }
        }
    }

    __device__ static void device_divide_and_round_q_last_ntt_step2(
        ConstSlice<Modulus> base_q, size_t coeff_count,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q, ConstSlice<uint64_t> input, 
        Slice<uint64_t> destination, size_t pcount, ConstSlice<uint64_t> temp
    ) {
        size_t j = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_q_size = base_q.size();
        if (j >= coeff_count) return;

        for (size_t p = 0; p < pcount; p++) {
            size_t poffset = p * coeff_count * base_q_size;
            size_t offset = p * coeff_count * (base_q_size - 1);
            for (size_t i = 0; i < base_q_size - 1; i++) {
                const Modulus& modulus = *base_q.at(i);
                uint64_t qi_lazy = modulus.value() << 2;
                uint64_t dest_ij = utils::add_uint64_mod(input[poffset + i * coeff_count + j], qi_lazy, modulus);
                dest_ij = utils::sub_uint64_mod(dest_ij, temp[offset + i * coeff_count + j], modulus);
                destination[offset + i * coeff_count + j] = utils::multiply_uint64operand_mod(dest_ij, inv_q_last_mod_q[i], modulus);
            }
        }
    }

    __global__ static void kernel_divide_and_round_q_last_ntt_step2(
        ConstSlice<Modulus> base_q, size_t coeff_count,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q, ConstSlice<uint64_t> input, 
        Slice<uint64_t> destination, size_t pcount, ConstSlice<uint64_t> temp
    ) {
        device_divide_and_round_q_last_ntt_step2(base_q, coeff_count, inv_q_last_mod_q, input, destination, pcount, temp);
    }

    __global__ static void kernel_divide_and_round_q_last_ntt_step2_batched(
        ConstSlice<Modulus> base_q, size_t coeff_count,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q, ConstSliceArrayRef<uint64_t> input, 
        SliceArrayRef<uint64_t> destination, size_t pcount, ConstSliceArrayRef<uint64_t> temp
    ) {
        size_t i = blockIdx.y;
        device_divide_and_round_q_last_ntt_step2(base_q, coeff_count, inv_q_last_mod_q, input[i], destination[i], pcount, temp[i]);
    }

    static void divide_and_round_q_last_ntt_step2(const RNSTool& self, ConstSlice<uint64_t> input, Slice<uint64_t> destination, size_t pcount, ConstSlice<uint64_t> temp) {
        bool device = self.on_device();
        if (device) {
            size_t block_count = utils::ceil_div(self.coeff_count(), utils::KERNEL_THREAD_COUNT);
            utils::set_device(destination.device_index());
            kernel_divide_and_round_q_last_ntt_step2<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                self.base_q().base(),
                self.coeff_count(),
                self.inv_q_last_mod_q(),
                input, destination, pcount,
                temp
            );
            utils::stream_sync();
        } else {
            host_divide_and_round_q_last_ntt_step2(self, input, destination, pcount, temp);
        }
    }
    
    void RNSTool::divide_and_round_q_last_ntt(ConstSlice<uint64_t> input, size_t pcount, Slice<uint64_t> destination, ConstSlice<NTTTables> rns_ntt_tables, MemoryPoolHandle pool) const {
        bool device = this->on_device();
        if (!utils::device_compatible(*this, input, rns_ntt_tables)) {
            throw std::invalid_argument("[RNSTool::divide_and_round_q_last_ntt_inplace] RNSTool, input and ntt_tables must be on the same device.");
        }

        size_t base_q_size = this->base_q().size();
        size_t coeff_count = this->coeff_count();

        Buffer<uint64_t> input_intt(pcount, base_q_size, coeff_count, device, pool);
        if (device) {
            // TODO: actually we only need the last component's intt, not all of them. This could be optimized.
            utils::intt_ps(input, pcount, coeff_count, rns_ntt_tables, input_intt.reference());
        } else {
            for (size_t i = 0; i < pcount; i++) {
                utils::intt(
                    input.const_slice((i * base_q_size + base_q_size - 1) * coeff_count, (i + 1) * base_q_size * coeff_count),
                    coeff_count,
                    rns_ntt_tables.at(base_q_size - 1),
                    input_intt.slice((i * base_q_size + base_q_size - 1) * coeff_count, (i + 1) * base_q_size * coeff_count)
                );
            }
        }

        Buffer<uint64_t> temp(pcount, base_q_size - 1, coeff_count, device, pool);
        divide_and_round_q_last_ntt_step1(*this, input_intt.reference(), pcount, temp.reference());
        
        utils::ntt_inplace_ps(temp.reference(), pcount, coeff_count, rns_ntt_tables.const_slice(0, base_q_size - 1));
    
        divide_and_round_q_last_ntt_step2(*this, input, destination, pcount, temp.const_reference());
    }

    void RNSTool::divide_and_round_q_last_ntt_batched(
        const ConstSliceVec<uint64_t>& input, size_t input_pcount, 
        const SliceVec<uint64_t>& destination, ConstSlice<NTTTables> rns_ntt_tables, 
        MemoryPoolHandle pool
    ) const {
        if (input.size() != destination.size()) {
            throw std::invalid_argument("[RNSTool::divide_and_round_q_last_ntt_batched] input and destination must have the same number of elements.");
        }
        if (input.size() == 0) return;
        bool device = this->on_device();
        size_t n = input.size();
        if (!device || n < BATCH_OP_THRESHOLD) {
            for (size_t i = 0; i < n; i++) {
                this->divide_and_round_q_last_ntt(input[i], input_pcount, destination[i], rns_ntt_tables, pool);
            }
        } else {
            std::vector<Buffer<uint64_t>> input_intt; input_intt.reserve(n);
            for (size_t i = 0; i < n; i++) {
                input_intt.push_back(Buffer<uint64_t>(input_pcount, this->base_q().size(), this->coeff_count(), device, pool));
            }
            size_t coeff_count = this->coeff_count();
            size_t base_q_size = this->base_q().size();
            utils::intt_bps(input, input_pcount, coeff_count, rns_ntt_tables, rcollect_reference(input_intt), pool);
            std::vector<Buffer<uint64_t>> temp; temp.reserve(n);
            for (size_t i = 0; i < n; i++) {
                temp.push_back(Buffer<uint64_t>(input_pcount, base_q_size - 1, coeff_count, device, pool));
            }
            size_t block_count = utils::ceil_div(coeff_count, utils::KERNEL_THREAD_COUNT);
            dim3 block_dims(block_count, n);
            utils::set_device(this->device_index());
            auto comp_ref = this->base_q().base();

            auto input_intt_batched = construct_batch(rcollect_reference(input_intt), pool, comp_ref);
            auto input_intt_const_batched = construct_batch(rcollect_const_reference(input_intt), pool, comp_ref);
            auto temp_batched = construct_batch(rcollect_reference(temp), pool, comp_ref);
            utils::set_device(comp_ref.device_index());
            kernel_divide_and_round_q_last_ntt_step1_batched<<<block_dims, utils::KERNEL_THREAD_COUNT>>>(
                this->base_q().base(),
                this->coeff_count(),
                this->q_last_half(),
                input_intt_const_batched,
                input_pcount,
                temp_batched
            );
            utils::stream_sync();

            utils::ntt_inplace_bps(rcollect_reference(temp), input_pcount, coeff_count, rns_ntt_tables.const_slice(0, base_q_size - 1), pool);

            auto input_batched = construct_batch(input, pool, comp_ref);
            auto destination_batched = construct_batch(destination, pool, comp_ref);
            auto temp_const_batched = construct_batch(rcollect_const_reference(temp), pool, comp_ref);
            utils::set_device(comp_ref.device_index());
            kernel_divide_and_round_q_last_ntt_step2_batched<<<block_dims, utils::KERNEL_THREAD_COUNT>>>(
                this->base_q().base(),
                this->coeff_count(),
                this->inv_q_last_mod_q(),
                input_batched,
                destination_batched,
                input_pcount,
                temp_const_batched
            );
            utils::stream_sync();

        }
    }

    static void host_fast_b_conv_sk_step1(const RNSTool& self, ConstSlice<uint64_t> input, Slice<uint64_t> destination, ConstSlice<uint64_t> temp) {
        size_t coeff_count = self.coeff_count();
        const Modulus& m_sk = *self.m_sk();
        uint64_t m_sk_value = m_sk.value();
        uint64_t m_sk_div_2 = m_sk_value >> 1;
        size_t base_B_size = self.base_B().size();
        for (size_t j = 0; j < coeff_count; j++) {
            uint64_t alpha_sk = multiply_uint64operand_mod(
                temp[j] + (m_sk_value - input[base_B_size * coeff_count + j]),
                self.inv_prod_B_mod_m_sk(),
                m_sk
            );
            for (size_t i = 0; i < self.base_q().size(); i++) {
                const Modulus& modulus = *self.base_q().base().at(i);
                MultiplyUint64Operand prod_B_mod_q_elt(self.prod_B_mod_q()[i], modulus);
                MultiplyUint64Operand neg_prod_B_mod_q_elt(modulus.value() - self.prod_B_mod_q()[i], modulus);
                uint64_t& dest = destination[i * coeff_count + j];
                if (alpha_sk > m_sk_div_2) {
                    dest = utils::multiply_uint64operand_add_uint64_mod(
                        utils::negate_uint64_mod(alpha_sk, m_sk), prod_B_mod_q_elt, dest, modulus
                    );
                } else {
                    dest = utils::multiply_uint64operand_add_uint64_mod(
                        alpha_sk, neg_prod_B_mod_q_elt, dest, modulus
                    );
                }
            }
        }
    }

    __global__ static void kernel_fast_b_conv_sk_step1(
        ConstSlice<Modulus> base_B,
        ConstSlice<Modulus> base_q,
        ConstPointer<Modulus> m_sk,
        MultiplyUint64Operand inv_prod_B_mod_m_sk,
        ConstSlice<uint64_t> prod_B_mod_q,
        size_t coeff_count,
        ConstSlice<uint64_t> input,
        Slice<uint64_t> destination,
        ConstSlice<uint64_t> temp
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_B_size = base_B.size();
        size_t base_q_size = base_q.size();
        if (global_index >= coeff_count * base_q_size) return;
        size_t i = global_index / coeff_count;
        size_t j = global_index % coeff_count;
        uint64_t m_sk_value = m_sk->value();
        uint64_t m_sk_div_2 = m_sk_value >> 1;
        uint64_t alpha_sk = multiply_uint64operand_mod(
            temp[j] + (m_sk_value - input[base_B_size * coeff_count + j]),
            inv_prod_B_mod_m_sk,
            *m_sk
        );
        const Modulus& modulus = *base_q.at(i);
        MultiplyUint64Operand prod_B_mod_q_elt(prod_B_mod_q[i], modulus);
        MultiplyUint64Operand neg_prod_B_mod_q_elt(modulus.value() - prod_B_mod_q[i], modulus);
        uint64_t& dest = destination[i * coeff_count + j];
        if (alpha_sk > m_sk_div_2) {
            dest = utils::multiply_uint64operand_add_uint64_mod(
                utils::negate_uint64_mod(alpha_sk, *m_sk), prod_B_mod_q_elt, dest, modulus
            );
        } else {
            dest = utils::multiply_uint64operand_add_uint64_mod(
                alpha_sk, neg_prod_B_mod_q_elt, dest, modulus
            );
        }
    }
    
    void RNSTool::fast_b_conv_sk(ConstSlice<uint64_t> input, Slice<uint64_t> destination, MemoryPoolHandle pool) const {
        bool device = this->on_device();
        if (!utils::device_compatible(*this, input, destination)) {
            throw std::invalid_argument("[RNSTool::fast_b_conv_sk] RNSTool, input and destination must be on the same device.");
        }
        size_t coeff_count = this->coeff_count();
        const RNSBase& base_B = this->base_B();
        size_t base_B_size = base_B.size();

        // Fast convert B -> q; input is in Bsk but we only use B
        this->base_B_to_q_conv().fast_convert_array(input.const_slice(0, base_B_size * coeff_count), destination, pool);
        
        // Compute alpha_sk
        // Fast convert B -> {m_sk}; input is in Bsk but we only use B
        Array<uint64_t> temp(coeff_count, device, pool);
        this->base_B_to_m_sk_conv().fast_convert_array(input.const_slice(0, base_B_size * coeff_count), temp.reference(), pool);
        
        if (device) {
            size_t base_q_size = this->base_q().size();
            size_t block_count = utils::ceil_div(coeff_count * base_q_size, utils::KERNEL_THREAD_COUNT);
            utils::set_device(input.device_index());
            kernel_fast_b_conv_sk_step1<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                base_B.base(),
                this->base_q().base(),
                this->m_sk(),
                this->inv_prod_B_mod_m_sk(),
                this->prod_B_mod_q(),
                coeff_count,
                input,
                destination,
                temp.const_reference()
            );
            utils::stream_sync();
        } else {
            host_fast_b_conv_sk_step1(*this, input, destination, temp.const_reference());
        }

    }

    static void host_sm_mrq(const RNSTool& self, ConstSlice<uint64_t> input, Slice<uint64_t> destination) {
        ConstSlice<Modulus> base_Bsk = self.base_Bsk().base();
        size_t base_Bsk_size = base_Bsk.size();
        size_t coeff_count = self.coeff_count();
        ConstSlice<uint64_t> input_m_tilde = input.const_slice(base_Bsk_size * coeff_count, (base_Bsk_size + 1) * coeff_count);
        uint64_t m_tilde_div_2 = self.m_tilde()->value() >> 1;
        Array<MultiplyUint64Operand> prod_q_mod_Bsk_elt(base_Bsk_size, false, nullptr);
        for (size_t i = 0; i < base_Bsk_size; i++) {
            const Modulus& modulus = *base_Bsk.at(i);
            prod_q_mod_Bsk_elt[i] = MultiplyUint64Operand(self.prod_q_mod_Bsk()[i], modulus);
        }
        for (size_t j = 0; j < coeff_count; j++) {
            uint64_t r_m_tilde = utils::multiply_uint64operand_mod(
                input_m_tilde[j], 
                self.neg_inv_prod_q_mod_m_tilde(),
                *self.m_tilde()
            );
            for (size_t i = 0; i < base_Bsk_size; i++) {
                const Modulus& modulus = *base_Bsk.at(i);
                uint64_t temp = r_m_tilde;
                if (temp >= m_tilde_div_2) {
                    temp += modulus.value() - self.m_tilde()->value();
                }
                destination[i * coeff_count + j] = utils::multiply_uint64operand_mod(
                    utils::multiply_uint64operand_add_uint64_mod(
                        temp,
                        prod_q_mod_Bsk_elt[i],
                        input[i * coeff_count + j],
                        modulus
                    ),
                    self.inv_m_tilde_mod_Bsk()[i],
                    modulus
                );
            }
        }
    }

    __global__ static void kernel_sm_mrq(
        ConstSlice<Modulus> base_Bsk,
        ConstPointer<Modulus> m_tilde,
        MultiplyUint64Operand neg_inv_prod_q_mod_m_tilde,
        ConstSlice<uint64_t> prod_q_mod_Bsk,
        ConstSlice<MultiplyUint64Operand> inv_m_tilde_mod_Bsk,
        size_t coeff_count,
        ConstSlice<uint64_t> input,
        Slice<uint64_t> destination
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_Bsk_size = base_Bsk.size();
        if (global_index >= coeff_count) return;
        size_t j = global_index % coeff_count;
        for (size_t i = 0; i < base_Bsk_size; i++) {
            const Modulus& modulus = *base_Bsk.at(i);
            uint64_t m_tilde_div_2 = m_tilde->value() >> 1;
            uint64_t r_m_tilde = utils::multiply_uint64operand_mod(
                input[base_Bsk_size * coeff_count + j], 
                neg_inv_prod_q_mod_m_tilde,
                *m_tilde
            );
            uint64_t temp = r_m_tilde;
            if (temp >= m_tilde_div_2) {
                temp += modulus.value() - m_tilde->value();
            }
            uint64_t& dest = destination[i * coeff_count + j];
            dest = utils::multiply_uint64operand_mod(
                utils::multiply_uint64operand_add_uint64_mod(
                    temp,
                    MultiplyUint64Operand(prod_q_mod_Bsk[i], modulus),
                    input[i * coeff_count + j],
                    modulus
                ),
                inv_m_tilde_mod_Bsk[i],
                modulus
            );
        }
    }
    
    void RNSTool::sm_mrq(ConstSlice<uint64_t> input, Slice<uint64_t> destination) const {
        bool device = this->on_device();
        if (!utils::device_compatible(*this, input, destination)) {
            throw std::invalid_argument("[RNSTool::sm_mrq] RNSTool, input and destination must be on the same device.");
        }
        size_t coeff_count = this->coeff_count();
        const RNSBase& base_Bsk = this->base_Bsk();
        if (device) {
            size_t block_count = utils::ceil_div(coeff_count, utils::KERNEL_THREAD_COUNT);
            utils::set_device(input.device_index());
            kernel_sm_mrq<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                base_Bsk.base(),
                this->m_tilde(),
                this->neg_inv_prod_q_mod_m_tilde(),
                this->prod_q_mod_Bsk(),
                this->inv_m_tilde_mod_Bsk(),
                coeff_count,
                input,
                destination
            );
            utils::stream_sync();
        } else {
            host_sm_mrq(*this, input, destination);
        }
    }

    static void host_fast_floor(const RNSTool& self, ConstSlice<uint64_t> input, Slice<uint64_t> destination) {
        size_t base_q_size = self.base_q().size();
        size_t base_Bsk_size = self.base_Bsk().size();
        size_t coeff_count = self.coeff_count();
        input = input.const_slice(base_q_size * coeff_count, input.size());
        for (size_t i = 0; i < base_Bsk_size; i++) {
            for (size_t j = 0; j < coeff_count; j++) {
                size_t index = i * coeff_count + j;
                destination[index] = utils::multiply_uint64operand_mod(
                    input[index] + self.base_Bsk().base()[i].value() - destination[index],
                    self.inv_prod_q_mod_Bsk()[i],
                    self.base_Bsk().base()[i]
                );
            }
        }
    }

    __global__ static void kernel_fast_floor(
        ConstSlice<Modulus> base_Bsk,
        ConstSlice<MultiplyUint64Operand> inv_prod_q_mod_Bsk,
        size_t coeff_count,
        size_t base_q_size,
        ConstSlice<uint64_t> input,
        Slice<uint64_t> destination
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_Bsk_size = base_Bsk.size();
        if (global_index >= coeff_count * base_Bsk_size) return;
        size_t i = global_index / coeff_count;
        uint64_t& dest = destination[global_index];
        dest = utils::multiply_uint64operand_mod(
            input[global_index + base_q_size * coeff_count] + base_Bsk[i].value() - dest,
            inv_prod_q_mod_Bsk[i],
            base_Bsk[i]
        );
    }

    void RNSTool::fast_floor(ConstSlice<uint64_t> input, Slice<uint64_t> destination, MemoryPoolHandle pool) const {
        size_t base_q_size = this->base_q().size();
        size_t base_Bsk_size = this->base_Bsk().size();
        size_t coeff_count = this->coeff_count();

        this->base_q_to_Bsk_conv().fast_convert_array(
            input.const_slice(0, base_q_size * coeff_count),
            destination, pool
        );

        if (this->on_device()) {
            size_t block_count = utils::ceil_div(coeff_count * base_Bsk_size, utils::KERNEL_THREAD_COUNT);
            utils::set_device(input.device_index());
            kernel_fast_floor<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                this->base_Bsk().base(),
                this->inv_prod_q_mod_Bsk(),
                coeff_count,
                base_q_size,
                input,
                destination
            );
            utils::stream_sync();
        } else {
            host_fast_floor(*this, input, destination);
        }

    }
    
    void RNSTool::fast_floor_fast_b_conv_sk(ConstSlice<uint64_t> input_q, ConstSlice<uint64_t> input_Bsk, Slice<uint64_t> destination, MemoryPoolHandle pool) const {
        if (!utils::device_compatible(*this, input_q, input_Bsk, destination)) {
            throw std::invalid_argument("[RNSTool::fast_floor_fast_b_conv_sk] RNSTool, input_q, input_Bsk and destination must be on the same device.");
        }
        bool device = this->on_device();
        size_t base_q_size = this->base_q().size();
        size_t base_Bsk_size = this->base_Bsk().size();
        size_t coeff_count = this->coeff_count();
        ConstSlice<Modulus> base_q = this->base_q().base();
        ConstSlice<Modulus> base_Bsk = this->base_Bsk().base();
        size_t dest_size = input_q.size() / base_q_size / coeff_count;
        if (!device) {
            Buffer<uint64_t> temp_q_Bsk(base_q_size + base_Bsk_size, coeff_count, device, pool);
            Buffer<uint64_t> temp_Bsk(base_Bsk_size, coeff_count, device, pool);
            uint64_t plain_modulus_value = this->t()->value();
            for (size_t i = 0; i < dest_size; i++) {
                // Bring together the base q and base Bsk components into a single allocation
                // Step (6): multiply base q components by t (plain_modulus)
                utils::multiply_scalar_p(
                    input_q.const_slice(i*coeff_count*base_q_size, (i+1)*coeff_count*base_q_size),
                    plain_modulus_value,
                    coeff_count,
                    base_q,
                    temp_q_Bsk.components(0, base_q_size)
                );
                utils::multiply_scalar_p(
                    input_Bsk.const_slice(i*coeff_count*base_Bsk_size, (i+1)*coeff_count*base_Bsk_size),
                    plain_modulus_value,
                    coeff_count,
                    base_Bsk,
                    temp_q_Bsk.components(base_q_size, base_q_size + base_Bsk_size)
                );
                // Step (7): divide by q and floor, producing a result in base Bsk
                this->fast_floor(temp_q_Bsk.const_reference(), temp_Bsk.reference(), pool);
                // Step (8): use Shenoy-Kumaresan method to convert the result to base q and write to encrypted1
                this->fast_b_conv_sk(temp_Bsk.const_reference(), destination.slice(i*coeff_count*base_q_size, (i+1)*coeff_count*base_q_size), pool);
            }
        } else {
            fgk::rns_tool::fast_floor_fast_b_conv_sk(
                input_q, input_Bsk, *this, dest_size, destination, pool
            );
        }
        
    }
    
    void RNSTool::fast_b_conv_m_tilde(ConstSlice<uint64_t> input, Slice<uint64_t> destination, MemoryPoolHandle pool) const {
        bool device = this->on_device();
        size_t base_q_size = this->base_q().size();
        size_t base_Bsk_size = this->base_Bsk().size();
        size_t coeff_count = this->coeff_count();
        Buffer<uint64_t> temp(base_q_size, coeff_count, device, pool);
        utils::multiply_scalar_p(input, this->m_tilde_value(), coeff_count, this->base_q().base(), temp.reference());
        this->base_q_to_Bsk_conv().fast_convert_array(temp.const_reference(),
            destination.slice(0, base_Bsk_size * coeff_count), pool);
        this->base_q_to_m_tilde_conv().fast_convert_array(temp.const_reference(),
            destination.slice(base_Bsk_size * coeff_count, (base_Bsk_size + 1) * coeff_count), pool);
    }

    void RNSTool::fast_b_conv_m_tilde_sm_mrq(ConstSlice<uint64_t> input, Slice<uint64_t> destination, MemoryPoolHandle pool) const {
        bool device = this->on_device();
        size_t base_Bsk_size = this->base_Bsk().size();
        size_t coeff_count = this->coeff_count();
        if (!input.on_device()) {
            Buffer<uint64_t> temp(base_Bsk_size + 1, coeff_count, device, pool);
            this->fast_b_conv_m_tilde(input, temp.reference(), pool);
            this->sm_mrq(temp.const_reference(), destination);
        } else {
            fgk::rns_tool::fast_b_conv_m_tilde_sm_mrq(
                input, coeff_count, this->m_tilde_value(), this->base_q().base(),
                this->base_q_to_Bsk_conv(), 
                this->base_q_to_m_tilde_conv(),
                this->neg_inv_prod_q_mod_m_tilde(),
                this->prod_q_mod_Bsk(),
                this->inv_m_tilde_mod_Bsk(),
                destination,
                pool
            );
        }
    }

    static void host_decrypt_scale_and_round_step1(const RNSTool& self, Slice<uint64_t> destination, size_t coeff_count, ConstSlice<uint64_t> temp_t_gamma) {
        uint64_t gamma = self.gamma()->value();
        uint64_t gamma_div_2 = gamma >> 1;
        const Modulus& t = *self.t();
        for (size_t i = 0; i < coeff_count; i++) {
            if (temp_t_gamma[coeff_count + i] > gamma_div_2) {
                destination[i] = add_uint64_mod(
                    temp_t_gamma[i], t.reduce(gamma - temp_t_gamma[coeff_count + i]), t
                );
            } else {
                destination[i] = sub_uint64_mod(
                    temp_t_gamma[i], t.reduce(temp_t_gamma[coeff_count + i]), t
                );
            }
            if (destination[i] != 0) {
                destination[i] = multiply_uint64operand_mod(destination[i], self.inv_gamma_mod_t(), t);
            }
        }
    }

    __global__ static void kernel_decrypt_scale_and_round_step1(
        ConstPointer<Modulus> gamma,
        ConstPointer<Modulus> t,
        MultiplyUint64Operand inv_gamma_mod_t,
        size_t coeff_count,
        Slice<uint64_t> destination,
        ConstSlice<uint64_t> temp_t_gamma
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= coeff_count) return;
        uint64_t gamma_value = gamma->value();
        uint64_t gamma_div_2 = gamma_value >> 1;
        uint64_t temp;
        uint64_t& dest = destination[i];
        if (temp_t_gamma[coeff_count + i] > gamma_div_2) {
            temp = add_uint64_mod(
                temp_t_gamma[i], t->reduce(gamma_value - temp_t_gamma[coeff_count + i]), *t
            );
        } else {
            temp = sub_uint64_mod(
                temp_t_gamma[i], t->reduce(temp_t_gamma[coeff_count + i]), *t
            );
        }
        if (temp != 0) {
            dest = multiply_uint64operand_mod(temp, inv_gamma_mod_t, *t);
        } else {
            dest = 0;
        }
    }

    static void decrypt_scale_and_round_step1(const RNSTool& self, Slice<uint64_t> destination, size_t coeff_count, ConstSlice<uint64_t> temp_t_gamma) {
        bool device = self.on_device();
        if (device) {
            size_t block_count = utils::ceil_div(coeff_count, utils::KERNEL_THREAD_COUNT);
            utils::set_device(destination.device_index());
            kernel_decrypt_scale_and_round_step1<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                self.gamma(),
                self.t(),
                self.inv_gamma_mod_t(),
                coeff_count,
                destination,
                temp_t_gamma
            );
            utils::stream_sync();
        } else {
            host_decrypt_scale_and_round_step1(self, destination, coeff_count, temp_t_gamma);
        }
    }

    // temp is (base_q_size + base_t_gamma_size) * coeff_count
    // fast_convert_temp is base_q_size * coeff_count
    __global__ void kernel_decrypt_scale_and_round_fused(
        ConstSlice<uint64_t> phase, size_t coeff_count, 
        ConstSlice<MultiplyUint64Operand> prod_t_gamma_mod_q, 
        ConstSlice<MultiplyUint64Operand> neg_inv_q_mod_t_gamma,
        ConstSlice<Modulus> base_q, ConstSlice<Modulus> base_t_gamma,

        ConstSlice<MultiplyUint64Operand> base_q_inv_punctured_product_mod_base,
        ConstSlice<uint64_t> fast_convert_base_change_matrix,

        ConstPointer<Modulus> gamma,
        ConstPointer<Modulus> t,
        MultiplyUint64Operand inv_gamma_mod_t,

        Slice<uint64_t> destination,

        Slice<uint64_t> temp,
        Slice<uint64_t> fast_convert_temp
    ) {
        size_t j = blockIdx.x * blockDim.x + threadIdx.x;
        if (j >= coeff_count) return;

        // multiply_uint64operand_p |gamma * t|_qi * ct(s)
        size_t base_q_size = base_q.size();
        for (size_t i = 0; i < base_q_size; i++) {
            temp[i * coeff_count + j] = multiply_uint64operand_mod(
                phase[i * coeff_count + j], prod_t_gamma_mod_q[i], base_q[i]
            );
        }

        // fast_convert_array
        size_t base_t_gamma_size = base_t_gamma.size();
        size_t offset = base_q_size * coeff_count;
        for (size_t i = 0; i < base_q_size; i++) {
            const MultiplyUint64Operand& op = base_q_inv_punctured_product_mod_base[i];
            const Modulus& base = base_q[i];
            if (op.operand == 1) {
                fast_convert_temp[j * base_q_size + i] = utils::barrett_reduce_uint64(temp[i * coeff_count + j], base);
            } else {
                fast_convert_temp[j * base_q_size + i] = utils::multiply_uint64operand_mod(temp[i * coeff_count + j], op, base);
            }
        }
        for (size_t i = 0; i < base_t_gamma_size; i++) {
            temp[offset + i * coeff_count + j] = utils::dot_product_mod(
                fast_convert_temp.const_slice(j * base_q_size, (j + 1) * base_q_size),
                fast_convert_base_change_matrix.const_slice(i * base_q_size, (i + 1) * base_q_size),
                base_t_gamma[i]
            );
        }
        
        // multiply_uint64operand_p -prod(q)^(-1) mod {t, gamma}
        for (size_t i = 0; i < base_t_gamma_size; i++) {
            temp[offset + i * coeff_count + j] = multiply_uint64operand_mod(
                temp[offset + i * coeff_count + j], neg_inv_q_mod_t_gamma[i], base_t_gamma[i]
            );
        }
        
        // Need to correct values in temp_t_gamma (gamma component only) which are
        // larger than floor(gamma/2)
        {
            uint64_t gamma_value = gamma->value();
            uint64_t gamma_div_2 = gamma_value >> 1;
            uint64_t temp_d;
            uint64_t& dest = destination[j];
            if (temp[offset + coeff_count + j] > gamma_div_2) {
                temp_d = add_uint64_mod(
                    temp[offset + j], t->reduce(gamma_value - temp[offset + coeff_count + j]), *t
                );
            } else {
                temp_d = sub_uint64_mod(
                    temp[offset + j], t->reduce(temp[offset + coeff_count + j]), *t
                );
            }
            if (temp_d != 0) {
                dest = multiply_uint64operand_mod(temp_d, inv_gamma_mod_t, *t);
            } else {
                dest = 0;
            }
        }

    }
    
    void RNSTool::decrypt_scale_and_round(ConstSlice<uint64_t> phase, size_t phase_coeff_count, Slice<uint64_t> destination, MemoryPoolHandle pool) const {
        bool device = this->on_device();
        if (!utils::device_compatible(*this, phase, destination)) {
            throw std::invalid_argument("[RNSTool::decrypt_scale_and_round] RNSTool, phase and destination must be on the same device.");
        }
        size_t base_q_size = this->base_q().size();
        size_t base_t_gamma_size = this->base_t_gamma().size();

        if (!device) {

            // Compute |gamma * t|_qi * ct(s)
            Array<uint64_t> temp = Array<uint64_t>::create_uninitialized(phase_coeff_count * base_q_size, device, pool);
            utils::multiply_uint64operand_p(
                phase.const_slice(0, base_q_size * phase_coeff_count),
                this->prod_t_gamma_mod_q(),
                phase_coeff_count,
                this->base_q().base(),
                temp.reference()
            );

            // Make another temp destination to get the poly in mod {t, gamma}
            Array<uint64_t> temp_t_gamma = Array<uint64_t>::create_uninitialized(phase_coeff_count * base_t_gamma_size, device, pool);
            this->base_q_to_t_gamma_conv()
                .fast_convert_array(temp.const_reference(), temp_t_gamma.reference(), pool);
            
            // Multiply by -prod(q)^(-1) mod {t, gamma}
            utils::multiply_uint64operand_inplace_p(
                temp_t_gamma.reference(),
                this->neg_inv_q_mod_t_gamma(),
                phase_coeff_count,
                this->base_t_gamma().base()
            );

            // Need to correct values in temp_t_gamma (gamma component only) which are
            // larger than floor(gamma/2)
            decrypt_scale_and_round_step1(*this, destination, phase_coeff_count, temp_t_gamma.const_reference());

        } else {

            Buffer<uint64_t> temp(base_q_size + base_t_gamma_size, phase_coeff_count, device, pool);
            Buffer<uint64_t> fast_convert_temp(base_q_size, phase_coeff_count, device, pool);
            size_t block_count = utils::ceil_div(phase_coeff_count, utils::KERNEL_THREAD_COUNT);
            utils::set_device(this->device_index());
            kernel_decrypt_scale_and_round_fused<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                phase, phase_coeff_count,
                this->prod_t_gamma_mod_q(),
                this->neg_inv_q_mod_t_gamma(),
                this->base_q().base(), this->base_t_gamma().base(),
                this->base_q_to_t_gamma_conv().input_base().inv_punctured_product_mod_base(),
                this->base_q_to_t_gamma_conv().base_change_matrix(),
                this->gamma(), this->t(), this->inv_gamma_mod_t(),
                destination,
                temp.reference(), fast_convert_temp.reference()
            );
            utils::stream_sync();

        }
    }

    static void host_mod_t_and_divide_q_last_inplace_step1(const RNSTool& self, Slice<uint64_t> input, ConstSlice<uint64_t> neg_c_last_mod_t) {
        if (self.on_device()) {
            throw std::logic_error("[host_mod_t_and_divide_q_last_inplace_step1] Unreachable.");
        }
        size_t base_q_size = self.base_q().size();
        size_t coeff_count = self.coeff_count();
        Array<uint64_t> delta_mod_q_i(coeff_count, false, nullptr);
        uint64_t last_modulus_value = self.base_q().base().at(base_q_size - 1)->value();
        for (size_t i = 0; i < base_q_size - 1; i++) {

            // delta_mod_q_i = neg_c_last_mod_t (mod q_i)
            ConstPointer<Modulus> modulus = self.base_q().base().at(i);
            utils::modulo(neg_c_last_mod_t, modulus, delta_mod_q_i.reference());

            // delta_mod_q_i *= q_last (mod q_i)
            utils::multiply_scalar_inplace(
                delta_mod_q_i.reference(), last_modulus_value, modulus
            );

            // c_i = c_i - c_last - neg_c_last_mod_t * q_last (mod 2q_i)
            uint64_t two_times_q_i = modulus->value() << 1;
            for (size_t j = 0; j < coeff_count; j++) {
                input[i * coeff_count + j] += two_times_q_i - modulus->reduce(
                    input[(base_q_size - 1) * coeff_count + j]
                ) - delta_mod_q_i[j];
            }
            
            // c_i = c_i * inv_q_last_mod_q_i (mod q_i)
            utils::multiply_uint64operand_inplace(
                input.slice(i * coeff_count, (i + 1) * coeff_count),
                self.inv_q_last_mod_q().at(i),
                modulus
            );
        }
    }

    __global__ static void kernel_mod_t_and_divide_q_last_inplace_step1(
        ConstSlice<Modulus> base_q,
        size_t coeff_count,
        ConstSlice<uint64_t> neg_c_last_mod_t,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q,
        Slice<uint64_t> input
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_q_size = base_q.size();
        if (global_index >= coeff_count * (base_q_size - 1)) return;
        size_t i = global_index / coeff_count;
        size_t j = global_index % coeff_count;

        uint64_t& dest = input[i * coeff_count + j];
        const Modulus& modulus = *base_q.at(i);
        uint64_t two_times_q_i = modulus.value() << 1;
        uint64_t delta_mod_q_i;
        // delta_mod_q_i = neg_c_last_mod_t (mod q_i)
        delta_mod_q_i = modulus.reduce(neg_c_last_mod_t[j]);
        // delta_mod_q_i *= q_last (mod q_i)
        delta_mod_q_i = utils::multiply_uint64_mod(delta_mod_q_i, base_q[base_q_size - 1].value(), modulus);
        // c_i = c_i - c_last - neg_c_last_mod_t * q_last (mod 2q_i)
        dest += two_times_q_i - modulus.reduce(input[(base_q_size - 1) * coeff_count + j]) - delta_mod_q_i;
        // c_i = c_i * inv_q_last_mod_q_i (mod q_i)
        dest = utils::multiply_uint64operand_mod(dest, inv_q_last_mod_q[i], modulus);
    }

    static void mod_t_and_divide_q_last_inplace_step1(const RNSTool& self, Slice<uint64_t> input, ConstSlice<uint64_t> neg_c_last_mod_t) {
        bool device = self.on_device();
        size_t base_q_size = self.base_q().size();
        size_t coeff_count = self.coeff_count();
        if (device) {
            size_t block_count = utils::ceil_div(coeff_count * (base_q_size - 1), utils::KERNEL_THREAD_COUNT);
            utils::set_device(input.device_index());
            kernel_mod_t_and_divide_q_last_inplace_step1<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                self.base_q().base(),
                coeff_count,
                neg_c_last_mod_t,
                self.inv_q_last_mod_q(),
                input
            );
            utils::stream_sync();
        } else {
            host_mod_t_and_divide_q_last_inplace_step1(self, input, neg_c_last_mod_t);
        }
    }

    void RNSTool::mod_t_and_divide_q_last_inplace(Slice<uint64_t> input, MemoryPoolHandle pool) const {
        bool device = this->on_device();
        if (!utils::device_compatible(*this, input)) {
            throw std::invalid_argument("[RNSTool::mod_t_and_divide_q_last_inplace] RNSTool and input must be on the same device.");
        }
        size_t modulus_size = this->base_q().size();
        size_t coeff_count = this->coeff_count();

        // neg_c_last_mod_t = - c_last (mod t)
        Array<uint64_t> neg_c_last_mod_t(coeff_count, device, pool);
        utils::modulo(
            input.const_slice((modulus_size - 1) * coeff_count, modulus_size * coeff_count),
            this->t(),
            neg_c_last_mod_t.reference()
        );
        utils::negate_inplace(neg_c_last_mod_t.reference(), this->t());
        if (this->inv_q_last_mod_t() != 1) {
            // neg_c_last_mod_t *= q_last^(-1) (mod t)
            utils::multiply_scalar_inplace(neg_c_last_mod_t.reference(), this->inv_q_last_mod_t(), this->t());
        }

        mod_t_and_divide_q_last_inplace_step1(*this, input, neg_c_last_mod_t.const_reference());

    }

    static void host_mod_t_and_divide_q_last_ntt(const RNSTool& self, ConstSlice<uint64_t> input, ConstSlice<uint64_t> input_intt, size_t pcount, Slice<uint64_t> destination, ConstSlice<NTTTables> rns_ntt_tables, MemoryPoolHandle pool) {
        bool device = self.on_device();
        size_t base_q_size = self.base_q().size();
        size_t coeff_count = self.coeff_count();
        Buffer<uint64_t> delta_mod_q_i(coeff_count, device, pool);
        uint64_t last_modulus_value = self.base_q().base().at(base_q_size - 1)->value();
        for (size_t p = 0; p < pcount; p++) {
            size_t poffset = p * base_q_size * coeff_count;
            size_t doffset = p * (base_q_size - 1) * coeff_count;
            Buffer<uint64_t> neg_c_last_mod_t(coeff_count, device, pool);
            utils::modulo(input_intt.const_slice(poffset + (base_q_size - 1) * coeff_count, poffset + base_q_size * coeff_count), self.t(), neg_c_last_mod_t.reference());
            utils::negate_inplace(neg_c_last_mod_t.reference(), self.t());
            if (self.inv_q_last_mod_t() != 1) {
                // neg_c_last_mod_t *= q_last^(-1) (mod t)
                utils::multiply_scalar_inplace(neg_c_last_mod_t.reference(), self.inv_q_last_mod_t(), self.t());
            }

            for (size_t i = 0; i < base_q_size - 1; i++) {

                // delta_mod_q_i = neg_c_last_mod_t (mod q_i)
                ConstPointer<Modulus> modulus = self.base_q().base().at(i);
                utils::modulo(neg_c_last_mod_t.const_reference(), modulus, delta_mod_q_i.reference());

                // delta_mod_q_i *= q_last (mod q_i)
                utils::multiply_scalar_inplace(
                    delta_mod_q_i.reference(), last_modulus_value, modulus
                );

                // c_i = c_i - c_last - neg_c_last_mod_t * q_last (mod 2q_i)
                //   first all all those to be subtracted to delta_mod_q_i
                for (size_t j = 0; j < coeff_count; j++) {
                    delta_mod_q_i[j] = add_uint64_mod(
                        delta_mod_q_i[j], 
                        modulus->reduce(input_intt[poffset + (base_q_size - 1) * coeff_count + j]),
                        *modulus
                    );
                }
                ntt_inplace(delta_mod_q_i.reference(), coeff_count, rns_ntt_tables.at(i));
                //   then subtract them all
                for (size_t j = 0; j < coeff_count; j++) {
                    destination[doffset + i * coeff_count + j] = sub_uint64_mod(
                        input[poffset + i * coeff_count + j], delta_mod_q_i[j], *modulus
                    );
                }
                
                // c_i = c_i * inv_q_last_mod_q_i (mod q_i)
                utils::multiply_uint64operand_inplace(
                    destination.slice(doffset + i * coeff_count, doffset + (i + 1) * coeff_count),
                    self.inv_q_last_mod_q().at(i),
                    modulus
                );
            }
        }
    }

    __device__ static void device_mod_t_and_divide_q_last_ntt_step1(
        ConstSlice<Modulus> base_q,
        ConstPointer<Modulus> t,
        size_t coeff_count,
        ConstSlice<uint64_t> input_intt,
        size_t pcount,
        uint64_t inv_q_last_mod_t,
        Slice<uint64_t> delta_mod_q_i
    ) {
        size_t j = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_q_size = base_q.size();
        if (j >= coeff_count) return;

        for (size_t p = 0; p < pcount; p++) {

            size_t poffset = p * base_q_size * coeff_count;
            size_t doffset = p * (base_q_size - 1) * coeff_count;

            uint64_t neg_c_last_mod_t = t->reduce(input_intt[poffset + (base_q_size - 1) * coeff_count + j]);
            neg_c_last_mod_t = utils::negate_uint64_mod(neg_c_last_mod_t, *t);
            if (inv_q_last_mod_t != 1) {
                // neg_c_last_mod_t *= q_last^(-1) (mod t)
                neg_c_last_mod_t = utils::multiply_uint64_mod(neg_c_last_mod_t, inv_q_last_mod_t, *t);
            }

            for (size_t i = 0; i < base_q.size() - 1; i++) {

                const Modulus& modulus = *base_q.at(i);
                uint64_t result;
                // delta_mod_q_i = neg_c_last_mod_t (mod q_i)
                result = modulus.reduce(neg_c_last_mod_t);
                // delta_mod_q_i *= q_last (mod q_i)
                result = utils::multiply_uint64_mod(result, base_q[base_q_size - 1].value(), modulus);
                // c_i = c_i - c_last - neg_c_last_mod_t * q_last (mod 2q_i)
                result = utils::add_uint64_mod(result, modulus.reduce(input_intt[poffset + (base_q_size - 1) * coeff_count + j]), modulus);
                delta_mod_q_i[doffset + i * coeff_count + j] = result;

            }
        }
    }

    __global__ static void kernel_mod_t_and_divide_q_last_ntt_step1(
        ConstSlice<Modulus> base_q,
        ConstPointer<Modulus> t,
        size_t coeff_count,
        ConstSlice<uint64_t> input_intt,
        size_t pcount,
        uint64_t inv_q_last_mod_t,
        Slice<uint64_t> delta_mod_q_i
    ) {
        device_mod_t_and_divide_q_last_ntt_step1(base_q, t, coeff_count, input_intt, pcount, inv_q_last_mod_t, delta_mod_q_i);
    }

    __global__ static void kernel_mod_t_and_divide_q_last_ntt_step1_batched(
        ConstSlice<Modulus> base_q,
        ConstPointer<Modulus> t,
        size_t coeff_count,
        ConstSliceArrayRef<uint64_t> input_intt,
        size_t pcount,
        uint64_t inv_q_last_mod_t,
        SliceArrayRef<uint64_t> delta_mod_q_i
    ) {
        size_t i = blockIdx.y;
        device_mod_t_and_divide_q_last_ntt_step1(base_q, t, coeff_count, input_intt[i], pcount, inv_q_last_mod_t, delta_mod_q_i[i]);
    }

    __device__ static void device_mod_t_and_divide_q_last_ntt_step2(
        ConstSlice<Modulus> base_q,
        size_t coeff_count,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q,
        ConstSlice<uint64_t> input,
        size_t pcount,
        Slice<uint64_t> destination,
        ConstSlice<uint64_t> delta_mod_q_i
    ) {
        size_t j = blockIdx.x * blockDim.x + threadIdx.x;
        size_t base_q_size = base_q.size();
        if (j >= coeff_count) return;

        for (size_t p = 0; p < pcount; p++) {
            size_t doffset = p * (base_q_size - 1) * coeff_count;
            size_t poffset = p * base_q_size * coeff_count;
            for (size_t i = 0; i < base_q.size() - 1; i++) {
                uint64_t dest = input[poffset + i * coeff_count + j];
                const Modulus& modulus = *base_q.at(i);
                // subtract
                dest = utils::sub_uint64_mod(dest, delta_mod_q_i[doffset + i * coeff_count + j], modulus);
                // c_i = c_i * inv_q_last_mod_q_i (mod q_i)
                dest = utils::multiply_uint64operand_mod(dest, inv_q_last_mod_q[i], modulus);
                destination[doffset + i * coeff_count + j] = dest;
            }
        }
    }

    __global__ static void kernel_mod_t_and_divide_q_last_ntt_step2(
        ConstSlice<Modulus> base_q,
        size_t coeff_count,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q,
        ConstSlice<uint64_t> input,
        size_t pcount,
        Slice<uint64_t> destination,
        ConstSlice<uint64_t> delta_mod_q_i
    ) {
        device_mod_t_and_divide_q_last_ntt_step2(base_q, coeff_count, inv_q_last_mod_q, input, pcount, destination, delta_mod_q_i);
    }
    __global__ static void kernel_mod_t_and_divide_q_last_ntt_step2_batched(
        ConstSlice<Modulus> base_q,
        size_t coeff_count,
        ConstSlice<MultiplyUint64Operand> inv_q_last_mod_q,
        ConstSliceArrayRef<uint64_t> input,
        size_t pcount,
        SliceArrayRef<uint64_t> destination,
        ConstSliceArrayRef<uint64_t> delta_mod_q_i
    ) {
        size_t i = blockIdx.y;
        device_mod_t_and_divide_q_last_ntt_step2(base_q, coeff_count, inv_q_last_mod_q, input[i], pcount, destination[i], delta_mod_q_i[i]);
    }

    static void mod_t_and_divide_q_last_ntt_step(const RNSTool& self, ConstSlice<uint64_t> input, ConstSlice<uint64_t> input_intt, size_t pcount, Slice<uint64_t> destination, ConstSlice<NTTTables> rns_ntt_tables, MemoryPoolHandle pool) {
        bool device = self.on_device();
        size_t base_q_size = self.base_q().size();
        size_t coeff_count = self.coeff_count();
        if (device) {
            size_t block_count = utils::ceil_div(coeff_count, utils::KERNEL_THREAD_COUNT);
            Buffer<uint64_t> delta_mod_q_i(pcount, base_q_size - 1, coeff_count, device, pool);
            utils::set_device(input.device_index());
            kernel_mod_t_and_divide_q_last_ntt_step1<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                self.base_q().base(),
                self.t(),
                coeff_count,
                input_intt, pcount, self.inv_q_last_mod_t(),
                delta_mod_q_i.reference()
            );
            utils::stream_sync();
            utils::ntt_inplace_ps(delta_mod_q_i.reference(), pcount, coeff_count, rns_ntt_tables.const_slice(0, base_q_size - 1));
            utils::set_device(input.device_index());
            kernel_mod_t_and_divide_q_last_ntt_step2<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                self.base_q().base(),
                coeff_count,
                self.inv_q_last_mod_q(),
                input,
                pcount,
                destination,
                delta_mod_q_i.const_reference()
            );
            utils::stream_sync();
        } else {
            host_mod_t_and_divide_q_last_ntt(self, input, input_intt, pcount, destination, rns_ntt_tables, pool);
        }
    }

    void RNSTool::mod_t_and_divide_q_last_ntt(ConstSlice<uint64_t> input, size_t pcount, Slice<uint64_t> destination, ConstSlice<NTTTables> rns_ntt_tables, MemoryPoolHandle pool) const {
        bool device = this->on_device();
        if (!utils::device_compatible(input, rns_ntt_tables, *this)) {
            throw std::invalid_argument("[RNSTool::mod_t_and_divide_q_last_ntt_inplace] RNSTool, input, rns_ntt_tables must be on the same device.");
        }
        
        size_t modulus_size = this->base_q().size();
        size_t coeff_count = this->coeff_count();

        Buffer<uint64_t> input_intt(pcount, modulus_size, coeff_count, device, pool);
        if (device) {
            // TODO: actually we only need the last component's intt, not all of them. This could be optimized.
            utils::intt_ps(input, pcount, coeff_count, rns_ntt_tables, input_intt.reference());
        } else {
            for (size_t i = 0; i < pcount; i++) {
                utils::intt(
                    input.const_slice((i * modulus_size + modulus_size - 1) * coeff_count, (i + 1) * modulus_size * coeff_count),
                    coeff_count,
                    rns_ntt_tables.at(modulus_size - 1),
                    input_intt.slice((i * modulus_size + modulus_size - 1) * coeff_count, (i + 1) * modulus_size * coeff_count)
                );
            }
        }

        mod_t_and_divide_q_last_ntt_step(*this, input, input_intt.const_reference(), pcount, destination, rns_ntt_tables, pool);

    }
    
    void RNSTool::mod_t_and_divide_q_last_ntt_batched(const ConstSliceVec<uint64_t>& input, size_t pcount, const SliceVec<uint64_t>& destination, ConstSlice<NTTTables> rns_ntt_tables, MemoryPoolHandle pool) const {
        if (input.size() != destination.size()) {
            throw std::invalid_argument("[RNSTool::mod_t_and_divide_q_last_ntt_batched] input and destination must have the same size.");
        }
        if (input.size() == 0) return;
        size_t n = input.size();
        bool device = this->on_device();
        if (!device || n < BATCH_OP_THRESHOLD) {
            for (size_t i = 0; i < n; i++) {
                this->mod_t_and_divide_q_last_ntt(input.at(i), pcount, destination.at(i), rns_ntt_tables, pool);
            }
        } else {
            size_t modulus_size = this->base_q().size();
            size_t coeff_count = this->coeff_count();
            std::vector<Buffer<uint64_t>> input_intt; input_intt.reserve(n);
            for (size_t i = 0; i < n; i++) {
                input_intt.emplace_back(pcount, this->base_q().size(), this->coeff_count(), device, pool);
            }
            utils::intt_bps(input, pcount, coeff_count, rns_ntt_tables, rcollect_reference(input_intt), pool);

            auto comp_ref = this->base_q().base();

            size_t block_count = utils::ceil_div(coeff_count, utils::KERNEL_THREAD_COUNT);
            dim3 block_dims(block_count, n);

            std::vector<Buffer<uint64_t>> delta_mod_q_i; delta_mod_q_i.reserve(n);
            for (size_t i = 0; i < n; i++) {
                delta_mod_q_i.emplace_back(pcount, modulus_size - 1, coeff_count, device, pool);
            }
            auto input_intt_const_batched = construct_batch(rcollect_const_reference(input_intt), pool, comp_ref);
            auto delta_mod_q_i_batched = construct_batch(rcollect_reference(delta_mod_q_i), pool, comp_ref);

            utils::set_device(comp_ref.device_index());
            kernel_mod_t_and_divide_q_last_ntt_step1_batched<<<block_dims, utils::KERNEL_THREAD_COUNT>>>(
                this->base_q().base(),
                this->t(),
                coeff_count,
                input_intt_const_batched, pcount, this->inv_q_last_mod_t(),
                delta_mod_q_i_batched
            );
            utils::stream_sync();
            utils::ntt_inplace_bps(rcollect_reference(delta_mod_q_i), pcount, coeff_count, rns_ntt_tables.const_slice(0, modulus_size - 1), pool);
            utils::set_device(comp_ref.device_index());

            auto input_batched = construct_batch(input, pool, comp_ref);
            auto delta_mod_q_i_const_batched = construct_batch(rcollect_const_reference(delta_mod_q_i), pool, comp_ref);
            auto destination_batched = construct_batch(destination, pool, comp_ref);
            kernel_mod_t_and_divide_q_last_ntt_step2_batched<<<block_dims, utils::KERNEL_THREAD_COUNT>>>(
                this->base_q().base(),
                coeff_count,
                this->inv_q_last_mod_q(),
                input_batched,
                pcount,
                destination_batched,
                delta_mod_q_i_const_batched
            );
            utils::stream_sync();

        }
    }

}}