#include "rlwe.h"

namespace troy {namespace rlwe {

    using utils::Array;
    using utils::ConstSlice;
    using utils::RandomGenerator;
    using utils::NTTTables;

    void asymmetric_with_u_prng(
        const PublicKey& pk,
        HeContextPointer context,
        const ParmsID& parms_id,
        bool is_ntt_form,
        RandomGenerator& u_prng,
        Ciphertext& destination,
        MemoryPoolHandle pool
    ) {

        destination.seed() = 0;

        bool device = pk.on_device();
        
        std::optional<ContextDataPointer> context_data_optional = context->get_context_data(parms_id);
        if (!context_data_optional.has_value()) {
            throw std::invalid_argument("[rlwe::asymmetric_with_u_prng] parms_id is not valid for the current context.");
        }
        ContextDataPointer context_data = context_data_optional.value();
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();
        const Ciphertext& public_key = pk.as_ciphertext();
        size_t encrypted_size = public_key.polynomial_count();
        SchemeType scheme_type = parms.scheme();
        RandomGenerator& context_prng = context->random_generator();

        // check device consistency
        if (context_data->on_device() != device) {
            throw std::invalid_argument("[rlwe::asymmetric_with_u_prng] context_data and public_key is not on the same device.");
        }

        // make destination have right size and parms_id
        if (device) destination.to_device_inplace(pool);
        else destination.to_host_inplace();
        destination.resize(context, parms_id, encrypted_size);
        destination.is_ntt_form() = is_ntt_form;
        destination.scale() = 1.0;
        destination.correction_factor() = 1;
        
        // c[j] = public_key[j] * u + e[j] in BFV/CKKS = public_key[j] * u + p * e[j] in BGV
        // where e[j] <-- chi, u <-- R_3

        // Create u <-- Ring_3
        Array<uint64_t> u(coeff_count * coeff_modulus_size, device, pool);
        u_prng.sample_poly_ternary(u.reference(), coeff_count, coeff_modulus);
        
        // c[j] = u * public_key[j]
        utils::ntt_inplace_p(u.reference(), coeff_count, ntt_tables);
        for (size_t j = 0; j < encrypted_size; j++) {
            utils::dyadic_product_p(
                u.const_reference(),
                public_key.poly(j),
                coeff_count, coeff_modulus, 
                destination.poly(j)
            );
        }
        if (!is_ntt_form) {
            utils::intt_inplace_ps(destination.data().reference(), encrypted_size, coeff_count, ntt_tables);
        }

        // Create e[j] <-- chi
        // c[j] = public_key[j] * u + e[j] in BFV/CKKS, = public_key[j] * u + p * e[j] in BGV,
        for (size_t j = 0; j < encrypted_size; j++) {
            // Reuse u as e
            context_prng.sample_poly_centered_binomial(u.reference(), coeff_count, coeff_modulus); 
            if (is_ntt_form) {
                utils::ntt_inplace_p(u.reference(), coeff_count, ntt_tables);
            }
            if (scheme_type == SchemeType::BGV) {
                utils::multiply_scalar_inplace_p(
                    u.reference(), parms.plain_modulus_host().value(), coeff_count, coeff_modulus
                );
            }
            utils::add_inplace_p(
                destination.poly(j), u.const_reference(), coeff_count, coeff_modulus
            );
        }
    }

    void asymmetric(
        const PublicKey& public_key,
        HeContextPointer context,
        const ParmsID& parms_id,
        bool is_ntt_form,
        Ciphertext& destination,
        MemoryPoolHandle pool
    ) {
        RandomGenerator& u_prng = context->random_generator();
        asymmetric_with_u_prng(
            public_key, context, parms_id, is_ntt_form, u_prng, destination, pool
        );
    }
    
    void symmetric_with_c1_prng(
        const SecretKey& sk,
        HeContextPointer context,
        const ParmsID& parms_id,
        bool is_ntt_form,
        utils::RandomGenerator& c1_prng,
        bool save_seed,
        Ciphertext& destination,
        MemoryPoolHandle pool
    ) {
        
        destination.seed() = 0;
        bool device = sk.on_device();
        
        std::optional<ContextDataPointer> context_data_optional = context->get_context_data(parms_id);
        if (!context_data_optional.has_value()) {
            throw std::invalid_argument("[rlwe::asymmetric_with_u_prng] parms_id is not valid for the current context.");
        }
        ContextDataPointer context_data = context_data_optional.value();
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();
        const Plaintext& secret_key = sk.as_plaintext();
        size_t encrypted_size = 2;
        SchemeType scheme_type = parms.scheme();
        RandomGenerator& context_prng = context->random_generator();

        // check device consistency
        if (context_data->on_device() != device) {
            throw std::invalid_argument("[rlwe::symmetric_with_c1_prng] context_data and public_key is not on the same device.");
        }

        // make destination have right size and parms_id
        if (device) destination.to_device_inplace(pool);
        else destination.to_host_inplace();

        destination.resize(context, parms_id, encrypted_size);
        destination.is_ntt_form() = is_ntt_form;
        destination.scale() = 1.0;
        destination.correction_factor() = 1;

        uint64_t seed = 0;
        while (seed == 0) seed = c1_prng.sample_uint64();

        RandomGenerator c1_new_prng(seed);
        
        // Generate ciphertext: (c[0], c[1]) = ([-(as+ e)]_q, a) in BFV/CKKS
        // Generate ciphertext: (c[0], c[1]) = ([-(as+pe)]_q, a) in BGV

        if (is_ntt_form || !save_seed) {
            // Directly sample NTT form
            c1_new_prng.sample_poly_uniform(destination.poly(1), coeff_count, coeff_modulus);
        } else if (save_seed) {
            // Sample non-NTT form and store the seed
            c1_new_prng.sample_poly_uniform(destination.poly(1), coeff_count, coeff_modulus);
            // Transform the c1 into NTT representation
            utils::ntt_inplace_p(destination.poly(1), coeff_count, ntt_tables);
        }
        if (save_seed) {
            destination.seed() = seed;
        }


        // Sample e <-- chi
        Array<uint64_t> noise(coeff_count * coeff_modulus_size, device, pool);
        context_prng.sample_poly_centered_binomial(noise.reference(), coeff_count, coeff_modulus);

        // Calculate -(as+ e) (mod q) and store in c[0] in BFV/CKKS
        // Calculate -(as+pe) (mod q) and store in c[0] in BGV
        utils::dyadic_product_p(
            secret_key.poly(), destination.poly(1).as_const(),
            coeff_count, coeff_modulus, destination.poly(0)
        );

        if (is_ntt_form) {
            // Transform the noise e into NTT representation
            utils::ntt_inplace_p(noise.reference(), coeff_count, ntt_tables);
        } else {
            utils::intt_inplace_p(destination.poly(0), coeff_count, ntt_tables);
        }
        if (scheme_type == SchemeType::BGV) {
            utils::multiply_scalar_inplace_p(
                noise.reference(), parms.plain_modulus_host().value(), coeff_count, coeff_modulus
            );
        }
        
        // c0 = as + noise
        utils::add_inplace_p(
            destination.poly(0), noise.const_reference(), coeff_count, coeff_modulus
        );
        
        // (as + noise, a) -> (-(as + noise), a),
        utils::negate_inplace_p(destination.poly(0), coeff_count, coeff_modulus);

        if (!is_ntt_form && !save_seed) {
            // Transform the c1 into non-NTT representation
            utils::intt_inplace_p(destination.poly(1), coeff_count, ntt_tables);
        }
    }

    void symmetric(
        const SecretKey& secret_key,
        HeContextPointer context,
        const ParmsID& parms_id,
        bool is_ntt_form,
        bool save_seed,
        Ciphertext& destination,
        MemoryPoolHandle pool
    ) {
        RandomGenerator& c1_prng = context->random_generator();
        symmetric_with_c1_prng(
            secret_key, context, parms_id, is_ntt_form, c1_prng, save_seed, destination, pool
        );
    }


}}