#include "hip/hip_runtime.h"
#include "batch_utils.h"
#include "ciphertext.h"
#include "encryption_parameters.h"
#include "evaluator.h"
#include "utils/dynamic_array.h"
#include "utils/memory_pool.h"
#include "utils/ntt.h"
#include "utils/polynomial_buffer.h"
#include "fgk/dyadic_convolute.h"
#include "fgk/translate_plain.h"
#include "evaluator_utils.h"

namespace troy {

    using utils::Slice;
    using utils::ConstSlice;
    using utils::NTTTables;
    using utils::RNSTool;
    using utils::Buffer;

    ContextDataPointer Evaluator::get_context_data(const char* prompt, const ParmsID& encrypted) const {
        auto context_data_ptr = context_->get_context_data(encrypted);
        if (!context_data_ptr.has_value()) {
            throw std::invalid_argument(std::string(prompt) + " Context data not found parms id.");
        }
        return context_data_ptr.value();
    }

    void Evaluator::bfv_multiply(const Ciphertext& encrypted1, const Ciphertext& encrypted2, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_not_ntt_form("[Evaluator::bfv_multiply_inplace]", encrypted1);
        check_is_not_ntt_form("[Evaluator::bfv_multiply_inplace]", encrypted2);

        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bfv_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> base_q = parms.coeff_modulus();
        size_t base_q_size = base_q.size();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        const RNSTool& rns_tool = context_data->rns_tool();
        ConstSlice<Modulus> base_Bsk = rns_tool.base_Bsk().base();
        size_t base_Bsk_size = base_Bsk.size();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;
        ConstSlice<NTTTables> base_q_ntt_tables = context_data->small_ntt_tables();
        ConstSlice<NTTTables> base_Bsk_ntt_tables = rns_tool.base_Bsk_ntt_tables();
        
        // Microsoft SEAL uses BEHZ-style RNS multiplication. This process is somewhat complex and consists of the
        // following steps:
        //
        // (1) Lift encrypted1 and encrypted2 (initially in base q) to an extended base q U Bsk U {m_tilde}
        // (2) Remove extra multiples of q from the results with Montgomery reduction, switching base to q U Bsk
        // (3) Transform the data to NTT form
        // (4) Compute the ciphertext polynomial product using dyadic multiplication
        // (5) Transform the data back from NTT form
        // (6) Multiply the result by t (plain_modulus)
        // (7) Scale the result by q using a divide-and-floor algorithm, switching base to Bsk
        // (8) Use Shenoy-Kumaresan method to convert the result to base q

        bool device = encrypted1.on_device();
        destination = Ciphertext::like(encrypted1, dest_size, false, pool);
        // Allocate space for a base q output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted1_q(encrypted1_size, base_q_size, coeff_count, device, pool);
        // Allocate space for a base Bsk output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted1_Bsk(encrypted1_size, base_Bsk_size, coeff_count, device, pool);

        // Perform BEHZ steps (1)-(3) for encrypted1
        // Make copy of input polynomial (in base q) and convert to NTT form
        utils::ntt_ps(encrypted1.const_reference(), encrypted1_size, coeff_count, base_q_ntt_tables, encrypted1_q.reference());
        // Allocate temporary space for a polynomial in the Bsk U {m_tilde} base
        for (size_t i = 0; i < encrypted1_size; i++) {
            // (1) Convert from base q to base Bsk U {m_tilde}
            rns_tool.fast_b_conv_m_tilde_sm_mrq(encrypted1.const_poly(i), encrypted1_Bsk.poly(i), pool);
        }
        // Transform to NTT form in base Bsk
        utils::ntt_inplace_ps(encrypted1_Bsk.reference(), encrypted1_size, coeff_count, base_Bsk_ntt_tables);

        // Repeat for encrypted2
        Buffer<uint64_t> encrypted2_q(encrypted2_size, base_q_size, coeff_count, device, pool);
        Buffer<uint64_t> encrypted2_Bsk(encrypted2_size, base_Bsk_size, coeff_count, device, pool);
        utils::ntt_ps(encrypted2.const_reference(), encrypted2_size, coeff_count, base_q_ntt_tables, encrypted2_q.reference());
        for (size_t i = 0; i < encrypted2_size; i++) {
            rns_tool.fast_b_conv_m_tilde_sm_mrq(encrypted2.poly(i), encrypted2_Bsk.poly(i), pool);
        }
        utils::ntt_inplace_ps(encrypted2_Bsk.reference(), encrypted2_size, coeff_count, base_Bsk_ntt_tables);

        // Allocate temporary space for the output of step (4)
        // We allocate space separately for the base q and the base Bsk components
        Buffer<uint64_t> temp_dest_q(dest_size, base_q_size, coeff_count, device, pool);
        Buffer<uint64_t> temp_dest_Bsk(dest_size, base_Bsk_size, coeff_count, device, pool);

        // Perform BEHZ step (4): dyadic multiplication on arbitrary size ciphertexts
        utils::fgk::dyadic_convolute::dyadic_convolute(
            encrypted1_q.const_reference(), encrypted2_q.const_reference(),
            encrypted1_size, encrypted2_size, base_q, coeff_count,
            temp_dest_q.reference(), pool
        );
        utils::fgk::dyadic_convolute::dyadic_convolute(
            encrypted1_Bsk.const_reference(), encrypted2_Bsk.const_reference(),
            encrypted1_size, encrypted2_size, base_Bsk, coeff_count,
            temp_dest_Bsk.reference(), pool
        );

        // Perform BEHZ step (5): transform data from NTT form
        // Lazy reduction here. The following multiplyPolyScalarCoeffmod will correct the value back to [0, p)
        utils::intt_inplace_ps(temp_dest_q.reference(), dest_size, coeff_count, base_q_ntt_tables);
        utils::intt_inplace_ps(temp_dest_Bsk.reference(), dest_size, coeff_count, base_Bsk_ntt_tables);

        // Perform BEHZ steps (6)-(8)
        rns_tool.fast_floor_fast_b_conv_sk(
            temp_dest_q.const_reference(), temp_dest_Bsk.const_reference(),
            destination.reference(), pool
        );
    }

    void Evaluator::ckks_multiply(const Ciphertext& encrypted1, const Ciphertext& encrypted2, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_ntt_form("[Evaluator::ckks_multiply_inplace]", encrypted1);
        check_is_ntt_form("[Evaluator::ckks_multiply_inplace]", encrypted2);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::ckks_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;
        destination = Ciphertext::like(encrypted1, dest_size, false, pool);

        utils::fgk::dyadic_convolute::dyadic_convolute(
            encrypted1.const_reference(), encrypted2.const_reference(),
            encrypted1_size, encrypted2_size, coeff_modulus, coeff_count,
            destination.data().reference(), pool
        );

        destination.scale() = encrypted1.scale() * encrypted2.scale();
        if (!is_scale_within_bounds(destination.scale(), context_data)) {
            throw std::invalid_argument("[Evaluator::ckks_multiply] Scale out of bounds");
        }
    }
    
    void Evaluator::bgv_multiply(const Ciphertext& encrypted1, const Ciphertext& encrypted2, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_ntt_form("[Evaluator::bgv_multiply]", encrypted1);
        check_is_ntt_form("[Evaluator::bgv_multiply]", encrypted2);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bgv_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;
        destination = Ciphertext::like(encrypted1, dest_size, false, pool);

        utils::fgk::dyadic_convolute::dyadic_convolute(
            encrypted1.const_reference(), encrypted2.const_reference(),
            encrypted1_size, encrypted2_size, coeff_modulus, coeff_count,
            destination.data().reference(), pool
        );

        destination.correction_factor() = utils::multiply_uint64_mod(
            encrypted1.correction_factor(),
            encrypted2.correction_factor(),
            parms.plain_modulus_host()
        );
    }
    
    void Evaluator::multiply(const Ciphertext& encrypted1, const Ciphertext& encrypted2, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::multiply]", encrypted1);
        check_no_seed("[Evaluator::multiply]", encrypted2);
        check_same_parms_id("[Evaluator::multiply]", encrypted1, encrypted2);
        SchemeType scheme = this->context()->key_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: {
                this->bfv_multiply(encrypted1, encrypted2, destination, pool);
                break;
            }
            case SchemeType::CKKS: {
                this->ckks_multiply(encrypted1, encrypted2, destination, pool);
                break;
            }
            case SchemeType::BGV: {
                this->bgv_multiply(encrypted1, encrypted2, destination, pool);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::multiply] Scheme not implemented.");
            }
        }
    }

    void Evaluator::bfv_square(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_not_ntt_form("[Evaluator::bfv_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bfv_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> base_q = parms.coeff_modulus();
        size_t base_q_size = base_q.size();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->multiply(encrypted, encrypted, destination, pool);
            return;
        }
        
        const RNSTool& rns_tool = context_data->rns_tool();
        ConstSlice<Modulus> base_Bsk = rns_tool.base_Bsk().base();
        size_t base_Bsk_size = base_Bsk.size();
        ConstSlice<Modulus> base_Bsk_m_tilde = rns_tool.base_Bsk_m_tilde().base();
        size_t base_Bsk_m_tilde_size = base_Bsk_m_tilde.size();
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;
        ConstSlice<NTTTables> base_q_ntt_tables = context_data->small_ntt_tables();
        ConstSlice<NTTTables> base_Bsk_ntt_tables = rns_tool.base_Bsk_ntt_tables();
        
        // Microsoft SEAL uses BEHZ-style RNS multiplication. This process is somewhat complex and consists of the
        // following steps:
        //
        // (1) Lift encrypted1 and encrypted2 (initially in base q) to an extended base q U Bsk U {m_tilde}
        // (2) Remove extra multiples of q from the results with Montgomery reduction, switching base to q U Bsk
        // (3) Transform the data to NTT form
        // (4) Compute the ciphertext polynomial product using dyadic multiplication
        // (5) Transform the data back from NTT form
        // (6) Multiply the result by t (plain_modulus)
        // (7) Scale the result by q using a divide-and-floor algorithm, switching base to Bsk
        // (8) Use Shenoy-Kumaresan method to convert the result to base q

        bool device = encrypted.on_device();
        destination = Ciphertext::like(encrypted, dest_size, false, pool);
        // Allocate space for a base q output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted_q(encrypted_size, base_q_size, coeff_count, device, pool);
        // Allocate space for a base Bsk output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted_Bsk(encrypted_size, base_Bsk_size, coeff_count, device, pool);

        // Perform BEHZ steps (1)-(3) for encrypted1
        // Make copy of input polynomial (in base q) and convert to NTT form
        utils::ntt_ps(encrypted.const_reference(), encrypted_size, coeff_count, base_q_ntt_tables, encrypted_q.reference());
        // Allocate temporary space for a polynomial in the Bsk U {m_tilde} base
        Buffer<uint64_t> temp(base_Bsk_m_tilde_size, coeff_count, device, pool);
        for (size_t i = 0; i < encrypted_size; i++) {
            // (1) Convert from base q to base Bsk U {m_tilde}
            rns_tool.fast_b_conv_m_tilde_sm_mrq(encrypted.const_poly(i), encrypted_Bsk.poly(i), pool);
        }
        // Transform to NTT form in base Bsk
        utils::ntt_inplace_ps(encrypted_Bsk.reference(), encrypted_size, coeff_count, base_Bsk_ntt_tables);

        // Allocate temporary space for the output of step (4)
        // We allocate space separately for the base q and the base Bsk components
        Buffer<uint64_t> temp_dest_q(dest_size, base_q_size, coeff_count, device, pool);
        Buffer<uint64_t> temp_dest_Bsk(dest_size, base_Bsk_size, coeff_count, device, pool);

        // Perform the BEHZ ciphertext square both for base q and base Bsk
        utils::fgk::dyadic_convolute::dyadic_square(
            encrypted_q.const_reference(), base_q, coeff_count,
            temp_dest_q.reference()
        );
        utils::fgk::dyadic_convolute::dyadic_square(
            encrypted_Bsk.const_reference(), base_Bsk, coeff_count,
            temp_dest_Bsk.reference()
        );
        
        // Perform BEHZ step (5): transform data from NTT form
        // Lazy reduction here. The following multiplyPolyScalarCoeffmod will correct the value back to [0, p)
        utils::intt_inplace_ps(temp_dest_q.reference(), dest_size, coeff_count, base_q_ntt_tables);
        utils::intt_inplace_ps(temp_dest_Bsk.reference(), dest_size, coeff_count, base_Bsk_ntt_tables);

        // Perform BEHZ steps (6)-(8)
        rns_tool.fast_floor_fast_b_conv_sk(
            temp_dest_q.const_reference(), temp_dest_Bsk.const_reference(),
            destination.reference(), pool
        );
    }

    void Evaluator::ckks_square(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_ntt_form("[Evaluator::ckks_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::ckks_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->multiply(encrypted, encrypted, destination, pool);
            return;
        }
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;
        destination = Ciphertext::like(encrypted, dest_size, false, pool);
        
        utils::fgk::dyadic_convolute::dyadic_square(
            encrypted.const_reference(), coeff_modulus, coeff_count,
            destination.data().reference()
        );

        destination.scale() = encrypted.scale() * encrypted.scale();
        if (!is_scale_within_bounds(destination.scale(), context_data)) {
            throw std::invalid_argument("[Evaluator::ckks_multiply_inplace] Scale out of bounds");
        }
    }
    
    void Evaluator::bgv_square(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_ntt_form("[Evaluator::bgv_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bgv_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->multiply(encrypted, encrypted, destination, pool);
            return;
        }
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;
        destination = Ciphertext::like(encrypted, dest_size, false, pool);

        utils::fgk::dyadic_convolute::dyadic_square(
            encrypted.const_reference(), coeff_modulus, coeff_count,
            destination.data().reference()
        );

        destination.correction_factor() = utils::multiply_uint64_mod(
            encrypted.correction_factor(),
            encrypted.correction_factor(),
            parms.plain_modulus_host()
        );
    }

    void Evaluator::square(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::square_inplace]", encrypted);
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: {
                this->bfv_square(encrypted, destination, pool);
                break;
            }
            case SchemeType::CKKS: {
                this->ckks_square(encrypted, destination, pool);
                break;
            }
            case SchemeType::BGV: {
                this->bgv_square(encrypted, destination, pool);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::square_inplace] Scheme not implemented.");
            }
        }
    }

}