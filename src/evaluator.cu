#include "hip/hip_runtime.h"
#include "batch_utils.h"
#include "ciphertext.h"
#include "encryption_parameters.h"
#include "evaluator.h"
#include "utils/dynamic_array.h"
#include "utils/memory_pool.h"
#include "utils/ntt.h"
#include "utils/polynomial_buffer.h"
#include "fgk/dyadic_convolute.h"
#include "fgk/translate_plain.h"
#include "evaluator_utils.h"

namespace troy {

    using utils::Slice;
    using utils::ConstSlice;
    using utils::NTTTables;
    using utils::RNSTool;
    using utils::Buffer;
    using utils::GaloisTool;

    ContextDataPointer Evaluator::get_context_data(const char* prompt, const ParmsID& encrypted) const {
        auto context_data_ptr = context_->get_context_data(encrypted);
        if (!context_data_ptr.has_value()) {
            throw std::invalid_argument(std::string(prompt) + " Context data not found parms id.");
        }
        return context_data_ptr.value();
    }

    void Evaluator::negate_inplace(Ciphertext& encrypted) const {
        check_ciphertext("[Evaluator::negate_inplace]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::negate_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t poly_count = encrypted.polynomial_count();
        size_t poly_degree = parms.poly_modulus_degree();
        utils::negate_inplace_ps(encrypted.data().reference(), poly_count, poly_degree, coeff_modulus);
    }

    void Evaluator::negate(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_ciphertext("[Evaluator::negate]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::negate]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        destination = Ciphertext::like(encrypted, false, pool);
        size_t poly_count = encrypted.polynomial_count();
        size_t poly_degree = parms.poly_modulus_degree();
        utils::negate_ps(encrypted.data().const_reference(), poly_count, poly_degree, coeff_modulus, destination.data().reference());
    }

    
    void Evaluator::negate_inplace_batched(const std::vector<Ciphertext*>& encrypted, MemoryPoolHandle pool) const {
        if (encrypted.size() == 0) return;
        check_ciphertext_vec("[Evaluator::negate_inplace_batched]", encrypted);
        ParmsID parms_id = get_vec_parms_id(encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::negate_inplace_batched]", parms_id);
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t poly_count = get_vec_polynomial_count(encrypted);
        size_t poly_degree = parms.poly_modulus_degree();
        utils::negate_inplace_bps(batch_utils::pcollect_reference(encrypted), poly_count, poly_degree, coeff_modulus, pool);
    }

    void Evaluator::negate_batched(const std::vector<const Ciphertext*>& encrypted, const std::vector<Ciphertext*>& destination, MemoryPoolHandle pool) const {
        if (encrypted.size() != destination.size()) {
            throw std::invalid_argument("[Evaluator::negate_batched] Size mismatch");
        }
        if (encrypted.size() == 0) return;
        check_ciphertext_vec("[Evaluator::negate_batched]", encrypted);
        ParmsID parms_id = get_vec_parms_id(encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::negate_batched]", parms_id);
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        for (size_t i = 0; i < encrypted.size(); i++) *destination[i] = Ciphertext::like(*encrypted[i], false, pool);
        size_t poly_count = get_vec_polynomial_count(encrypted);
        size_t poly_degree = parms.poly_modulus_degree();
        utils::negate_bps(batch_utils::pcollect_const_reference(encrypted), poly_count, poly_degree, coeff_modulus, batch_utils::pcollect_reference(destination));
    }

    void Evaluator::bfv_multiply(const Ciphertext& encrypted1, const Ciphertext& encrypted2, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_not_ntt_form("[Evaluator::bfv_multiply_inplace]", encrypted1);
        check_is_not_ntt_form("[Evaluator::bfv_multiply_inplace]", encrypted2);

        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bfv_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> base_q = parms.coeff_modulus();
        size_t base_q_size = base_q.size();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        const RNSTool& rns_tool = context_data->rns_tool();
        ConstSlice<Modulus> base_Bsk = rns_tool.base_Bsk().base();
        size_t base_Bsk_size = base_Bsk.size();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;
        ConstSlice<NTTTables> base_q_ntt_tables = context_data->small_ntt_tables();
        ConstSlice<NTTTables> base_Bsk_ntt_tables = rns_tool.base_Bsk_ntt_tables();
        
        // Microsoft SEAL uses BEHZ-style RNS multiplication. This process is somewhat complex and consists of the
        // following steps:
        //
        // (1) Lift encrypted1 and encrypted2 (initially in base q) to an extended base q U Bsk U {m_tilde}
        // (2) Remove extra multiples of q from the results with Montgomery reduction, switching base to q U Bsk
        // (3) Transform the data to NTT form
        // (4) Compute the ciphertext polynomial product using dyadic multiplication
        // (5) Transform the data back from NTT form
        // (6) Multiply the result by t (plain_modulus)
        // (7) Scale the result by q using a divide-and-floor algorithm, switching base to Bsk
        // (8) Use Shenoy-Kumaresan method to convert the result to base q

        bool device = encrypted1.on_device();
        destination = Ciphertext::like(encrypted1, dest_size, false, pool);
        // Allocate space for a base q output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted1_q(encrypted1_size, base_q_size, coeff_count, device, pool);
        // Allocate space for a base Bsk output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted1_Bsk(encrypted1_size, base_Bsk_size, coeff_count, device, pool);

        // Perform BEHZ steps (1)-(3) for encrypted1
        // Make copy of input polynomial (in base q) and convert to NTT form
        utils::ntt_ps(encrypted1.const_reference(), encrypted1_size, coeff_count, base_q_ntt_tables, encrypted1_q.reference());
        // Allocate temporary space for a polynomial in the Bsk U {m_tilde} base
        for (size_t i = 0; i < encrypted1_size; i++) {
            // (1) Convert from base q to base Bsk U {m_tilde}
            rns_tool.fast_b_conv_m_tilde_sm_mrq(encrypted1.const_poly(i), encrypted1_Bsk.poly(i), pool);
        }
        // Transform to NTT form in base Bsk
        utils::ntt_inplace_ps(encrypted1_Bsk.reference(), encrypted1_size, coeff_count, base_Bsk_ntt_tables);

        // Repeat for encrypted2
        Buffer<uint64_t> encrypted2_q(encrypted2_size, base_q_size, coeff_count, device, pool);
        Buffer<uint64_t> encrypted2_Bsk(encrypted2_size, base_Bsk_size, coeff_count, device, pool);
        utils::ntt_ps(encrypted2.const_reference(), encrypted2_size, coeff_count, base_q_ntt_tables, encrypted2_q.reference());
        for (size_t i = 0; i < encrypted2_size; i++) {
            rns_tool.fast_b_conv_m_tilde_sm_mrq(encrypted2.poly(i), encrypted2_Bsk.poly(i), pool);
        }
        utils::ntt_inplace_ps(encrypted2_Bsk.reference(), encrypted2_size, coeff_count, base_Bsk_ntt_tables);

        // Allocate temporary space for the output of step (4)
        // We allocate space separately for the base q and the base Bsk components
        Buffer<uint64_t> temp_dest_q(dest_size, base_q_size, coeff_count, device, pool);
        Buffer<uint64_t> temp_dest_Bsk(dest_size, base_Bsk_size, coeff_count, device, pool);

        // Perform BEHZ step (4): dyadic multiplication on arbitrary size ciphertexts
        utils::fgk::dyadic_convolute::dyadic_convolute(
            encrypted1_q.const_reference(), encrypted2_q.const_reference(),
            encrypted1_size, encrypted2_size, base_q, coeff_count,
            temp_dest_q.reference(), pool
        );
        utils::fgk::dyadic_convolute::dyadic_convolute(
            encrypted1_Bsk.const_reference(), encrypted2_Bsk.const_reference(),
            encrypted1_size, encrypted2_size, base_Bsk, coeff_count,
            temp_dest_Bsk.reference(), pool
        );

        // Perform BEHZ step (5): transform data from NTT form
        // Lazy reduction here. The following multiplyPolyScalarCoeffmod will correct the value back to [0, p)
        utils::intt_inplace_ps(temp_dest_q.reference(), dest_size, coeff_count, base_q_ntt_tables);
        utils::intt_inplace_ps(temp_dest_Bsk.reference(), dest_size, coeff_count, base_Bsk_ntt_tables);

        // Perform BEHZ steps (6)-(8)
        rns_tool.fast_floor_fast_b_conv_sk(
            temp_dest_q.const_reference(), temp_dest_Bsk.const_reference(),
            destination.reference(), pool
        );
    }

    void Evaluator::ckks_multiply(const Ciphertext& encrypted1, const Ciphertext& encrypted2, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_ntt_form("[Evaluator::ckks_multiply_inplace]", encrypted1);
        check_is_ntt_form("[Evaluator::ckks_multiply_inplace]", encrypted2);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::ckks_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;
        destination = Ciphertext::like(encrypted1, dest_size, false, pool);

        utils::fgk::dyadic_convolute::dyadic_convolute(
            encrypted1.const_reference(), encrypted2.const_reference(),
            encrypted1_size, encrypted2_size, coeff_modulus, coeff_count,
            destination.data().reference(), pool
        );

        destination.scale() = encrypted1.scale() * encrypted2.scale();
        if (!is_scale_within_bounds(destination.scale(), context_data)) {
            throw std::invalid_argument("[Evaluator::ckks_multiply] Scale out of bounds");
        }
    }
    
    void Evaluator::bgv_multiply(const Ciphertext& encrypted1, const Ciphertext& encrypted2, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_ntt_form("[Evaluator::bgv_multiply]", encrypted1);
        check_is_ntt_form("[Evaluator::bgv_multiply]", encrypted2);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bgv_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;
        destination = Ciphertext::like(encrypted1, dest_size, false, pool);

        utils::fgk::dyadic_convolute::dyadic_convolute(
            encrypted1.const_reference(), encrypted2.const_reference(),
            encrypted1_size, encrypted2_size, coeff_modulus, coeff_count,
            destination.data().reference(), pool
        );

        destination.correction_factor() = utils::multiply_uint64_mod(
            encrypted1.correction_factor(),
            encrypted2.correction_factor(),
            parms.plain_modulus_host()
        );
    }
    
    void Evaluator::multiply(const Ciphertext& encrypted1, const Ciphertext& encrypted2, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::multiply]", encrypted1);
        check_no_seed("[Evaluator::multiply]", encrypted2);
        check_same_parms_id("[Evaluator::multiply]", encrypted1, encrypted2);
        SchemeType scheme = this->context()->key_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: {
                this->bfv_multiply(encrypted1, encrypted2, destination, pool);
                break;
            }
            case SchemeType::CKKS: {
                this->ckks_multiply(encrypted1, encrypted2, destination, pool);
                break;
            }
            case SchemeType::BGV: {
                this->bgv_multiply(encrypted1, encrypted2, destination, pool);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::multiply] Scheme not implemented.");
            }
        }
    }

    void Evaluator::bfv_square(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_not_ntt_form("[Evaluator::bfv_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bfv_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> base_q = parms.coeff_modulus();
        size_t base_q_size = base_q.size();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->multiply(encrypted, encrypted, destination, pool);
            return;
        }
        
        const RNSTool& rns_tool = context_data->rns_tool();
        ConstSlice<Modulus> base_Bsk = rns_tool.base_Bsk().base();
        size_t base_Bsk_size = base_Bsk.size();
        ConstSlice<Modulus> base_Bsk_m_tilde = rns_tool.base_Bsk_m_tilde().base();
        size_t base_Bsk_m_tilde_size = base_Bsk_m_tilde.size();
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;
        ConstSlice<NTTTables> base_q_ntt_tables = context_data->small_ntt_tables();
        ConstSlice<NTTTables> base_Bsk_ntt_tables = rns_tool.base_Bsk_ntt_tables();
        
        // Microsoft SEAL uses BEHZ-style RNS multiplication. This process is somewhat complex and consists of the
        // following steps:
        //
        // (1) Lift encrypted1 and encrypted2 (initially in base q) to an extended base q U Bsk U {m_tilde}
        // (2) Remove extra multiples of q from the results with Montgomery reduction, switching base to q U Bsk
        // (3) Transform the data to NTT form
        // (4) Compute the ciphertext polynomial product using dyadic multiplication
        // (5) Transform the data back from NTT form
        // (6) Multiply the result by t (plain_modulus)
        // (7) Scale the result by q using a divide-and-floor algorithm, switching base to Bsk
        // (8) Use Shenoy-Kumaresan method to convert the result to base q

        bool device = encrypted.on_device();
        destination = Ciphertext::like(encrypted, dest_size, false, pool);
        // Allocate space for a base q output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted_q(encrypted_size, base_q_size, coeff_count, device, pool);
        // Allocate space for a base Bsk output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted_Bsk(encrypted_size, base_Bsk_size, coeff_count, device, pool);

        // Perform BEHZ steps (1)-(3) for encrypted1
        // Make copy of input polynomial (in base q) and convert to NTT form
        utils::ntt_ps(encrypted.const_reference(), encrypted_size, coeff_count, base_q_ntt_tables, encrypted_q.reference());
        // Allocate temporary space for a polynomial in the Bsk U {m_tilde} base
        Buffer<uint64_t> temp(base_Bsk_m_tilde_size, coeff_count, device, pool);
        for (size_t i = 0; i < encrypted_size; i++) {
            // (1) Convert from base q to base Bsk U {m_tilde}
            rns_tool.fast_b_conv_m_tilde_sm_mrq(encrypted.const_poly(i), encrypted_Bsk.poly(i), pool);
        }
        // Transform to NTT form in base Bsk
        utils::ntt_inplace_ps(encrypted_Bsk.reference(), encrypted_size, coeff_count, base_Bsk_ntt_tables);

        // Allocate temporary space for the output of step (4)
        // We allocate space separately for the base q and the base Bsk components
        Buffer<uint64_t> temp_dest_q(dest_size, base_q_size, coeff_count, device, pool);
        Buffer<uint64_t> temp_dest_Bsk(dest_size, base_Bsk_size, coeff_count, device, pool);

        // Perform the BEHZ ciphertext square both for base q and base Bsk
        utils::fgk::dyadic_convolute::dyadic_square(
            encrypted_q.const_reference(), base_q, coeff_count,
            temp_dest_q.reference()
        );
        utils::fgk::dyadic_convolute::dyadic_square(
            encrypted_Bsk.const_reference(), base_Bsk, coeff_count,
            temp_dest_Bsk.reference()
        );
        
        // Perform BEHZ step (5): transform data from NTT form
        // Lazy reduction here. The following multiplyPolyScalarCoeffmod will correct the value back to [0, p)
        utils::intt_inplace_ps(temp_dest_q.reference(), dest_size, coeff_count, base_q_ntt_tables);
        utils::intt_inplace_ps(temp_dest_Bsk.reference(), dest_size, coeff_count, base_Bsk_ntt_tables);

        // Perform BEHZ steps (6)-(8)
        rns_tool.fast_floor_fast_b_conv_sk(
            temp_dest_q.const_reference(), temp_dest_Bsk.const_reference(),
            destination.reference(), pool
        );
    }

    void Evaluator::ckks_square(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_ntt_form("[Evaluator::ckks_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::ckks_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->multiply(encrypted, encrypted, destination, pool);
            return;
        }
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;
        destination = Ciphertext::like(encrypted, dest_size, false, pool);
        
        utils::fgk::dyadic_convolute::dyadic_square(
            encrypted.const_reference(), coeff_modulus, coeff_count,
            destination.data().reference()
        );

        destination.scale() = encrypted.scale() * encrypted.scale();
        if (!is_scale_within_bounds(destination.scale(), context_data)) {
            throw std::invalid_argument("[Evaluator::ckks_multiply_inplace] Scale out of bounds");
        }
    }
    
    void Evaluator::bgv_square(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_is_ntt_form("[Evaluator::bgv_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bgv_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->multiply(encrypted, encrypted, destination, pool);
            return;
        }
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;
        destination = Ciphertext::like(encrypted, dest_size, false, pool);

        utils::fgk::dyadic_convolute::dyadic_square(
            encrypted.const_reference(), coeff_modulus, coeff_count,
            destination.data().reference()
        );

        destination.correction_factor() = utils::multiply_uint64_mod(
            encrypted.correction_factor(),
            encrypted.correction_factor(),
            parms.plain_modulus_host()
        );
    }

    void Evaluator::square(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::square_inplace]", encrypted);
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: {
                this->bfv_square(encrypted, destination, pool);
                break;
            }
            case SchemeType::CKKS: {
                this->ckks_square(encrypted, destination, pool);
                break;
            }
            case SchemeType::BGV: {
                this->bgv_square(encrypted, destination, pool);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::square_inplace] Scheme not implemented.");
            }
        }
    }

    void Evaluator::apply_keyswitching(const Ciphertext& encrypted, const KSwitchKeys& kswitch_keys, Ciphertext& destination, MemoryPoolHandle pool) const {
        if (kswitch_keys.data().size() != 1) {
            throw std::invalid_argument("[Evaluator::apply_keyswitching_inplace] Key switch keys size must be 1.");
        }
        if (encrypted.polynomial_count() != 2) {
            throw std::invalid_argument("[Evaluator::apply_keyswitching_inplace] Ciphertext polynomial count must be 2.");
        }
        if (kswitch_keys.data()[0][0].as_ciphertext().polynomial_count() != 2) {
            throw std::invalid_argument("[Evaluator::apply_keyswitching_inplace] Key switch keys polynomial count must be 2. Check the key switch key generation for problems.");
        }
        destination = Ciphertext::like(encrypted, false, pool);
        this->switch_key_internal(encrypted, encrypted.poly(1), kswitch_keys, 0, Evaluator::SwitchKeyDestinationAssignMethod::Overwrite, destination, pool);
        
        ContextDataPointer context_data = this->get_context_data("[Evaluator::switch_key_inplace_internal]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();

        utils::add_inplace_p(destination.poly(0), encrypted.poly(0), parms.poly_modulus_degree(), parms.coeff_modulus());
    }

    void Evaluator::apply_keyswitching_inplace(Ciphertext& encrypted, const KSwitchKeys& kswitch_keys, MemoryPoolHandle pool) const {
        if (kswitch_keys.data().size() != 1) {
            throw std::invalid_argument("[Evaluator::apply_keyswitching_inplace] Key switch keys size must be 1.");
        }
        if (encrypted.polynomial_count() != 2) {
            throw std::invalid_argument("[Evaluator::apply_keyswitching_inplace] Ciphertext polynomial count must be 2.");
        }
        if (kswitch_keys.data()[0][0].as_ciphertext().polynomial_count() != 2) {
            throw std::invalid_argument("[Evaluator::apply_keyswitching_inplace] Key switch keys polynomial count must be 2. Check the key switch key generation for problems.");
        }
        this->switch_key_internal(encrypted, encrypted.poly(1), kswitch_keys, 0, Evaluator::SwitchKeyDestinationAssignMethod::OverwriteExceptFirst, encrypted, pool);
        
    }

    void Evaluator::relinearize_inplace_internal(Ciphertext& encrypted, const RelinKeys& relin_keys, size_t destination_size, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::relinearize_inplace_internal]", encrypted);
        if (relin_keys.parms_id() != this->context()->key_parms_id()) {
            throw std::invalid_argument("[Evaluator::relinearize_inplace_internal] Relin keys has incorrect parms id.");
        }
        ContextDataPointer context_data = this->get_context_data("[Evaluator::relinearize_inplace_internal]", encrypted.parms_id());
        size_t encrypted_size = encrypted.polynomial_count();
        if (encrypted_size < 2 || destination_size > encrypted_size) {
            throw std::invalid_argument("[Evaluator::relinearize_inplace_internal] Destination size must be at least 2 and less/equal to the size of the encrypted polynomial.");
        }
        if (destination_size == encrypted_size) {
            return;
        }
        size_t relins_needed = encrypted_size - destination_size;
        for (size_t i = 0; i < relins_needed; i++) {
            this->switch_key_internal(
                encrypted, encrypted.const_poly(encrypted_size - 1),
                relin_keys.as_kswitch_keys(), RelinKeys::get_index(encrypted_size - 1), Evaluator::SwitchKeyDestinationAssignMethod::AddInplace, encrypted, pool);
            encrypted_size -= 1;
        }
        encrypted.resize(this->context(), context_data->parms_id(), destination_size);
    }

    void Evaluator::relinearize_internal(const Ciphertext& encrypted, const RelinKeys& relin_keys, size_t destination_size, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::relinearize_inplace_internal]", encrypted);
        if (relin_keys.parms_id() != this->context()->key_parms_id()) {
            throw std::invalid_argument("[Evaluator::relinearize_inplace_internal] Relin keys has incorrect parms id.");
        }
        ContextDataPointer context_data = this->get_context_data("[Evaluator::relinearize_inplace_internal]", encrypted.parms_id());
        size_t encrypted_size = encrypted.polynomial_count();
        if (encrypted_size < 2 || destination_size > encrypted_size) {
            throw std::invalid_argument("[Evaluator::relinearize_inplace_internal] Destination size must be at least 2 and less/equal to the size of the encrypted polynomial.");
        }
        if (destination_size == encrypted_size) {
            return;
        }
        size_t relins_needed = encrypted_size - destination_size;
        destination = Ciphertext::like(encrypted, destination_size, false, pool);
        for (size_t i = 0; i < relins_needed; i++) {
            this->switch_key_internal(
                encrypted, encrypted.const_poly(encrypted_size - 1),
                relin_keys.as_kswitch_keys(), RelinKeys::get_index(encrypted_size - 1), 
                i == 0 ? Evaluator::SwitchKeyDestinationAssignMethod::Overwrite : Evaluator::SwitchKeyDestinationAssignMethod::AddInplace, 
                destination, pool);
            encrypted_size -= 1;
        }
        const EncryptionParameters& parms = context_data->parms();
        utils::add_inplace_ps(destination.polys(0, destination_size), encrypted.const_polys(0, destination_size), destination_size, parms.poly_modulus_degree(), parms.coeff_modulus());
    }

    void Evaluator::mod_switch_scale_to_next_internal(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        ParmsID parms_id = encrypted.parms_id();
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_scale_to_next_internal]", parms_id);
        const EncryptionParameters& parms = context_data->parms();
        SchemeType scheme = parms.scheme();
        switch (scheme) {
            case SchemeType::BFV: {
                check_is_not_ntt_form("[Evaluator::mod_switch_scale_to_next_internal]", encrypted);
                break;
            }
            case SchemeType::CKKS: case SchemeType::BGV: {
                check_is_ntt_form("[Evaluator::mod_switch_scale_to_next_internal]", encrypted);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::mod_switch_scale_to_next_internal] Scheme not implemented.");
            }
        }
        if (!context_data->next_context_data().has_value()) {
            throw std::invalid_argument("[Evaluator::mod_switch_scale_to_next_internal] Next context data is not set.");
        }
        ContextDataPointer next_context_data = context_data->next_context_data().value();
        const EncryptionParameters& next_parms = next_context_data->parms();
        const RNSTool& rns_tool = context_data->rns_tool();
        
        size_t encrypted_size = encrypted.polynomial_count();

        bool device = encrypted.on_device();
        if (device) destination.to_device_inplace(pool);
        else destination.to_host_inplace();
        destination.resize(this->context(), next_context_data->parms_id(), encrypted_size, false);

        switch (scheme) {
            case SchemeType::BFV: {
                rns_tool.divide_and_round_q_last(encrypted.reference(), encrypted_size, destination.reference());
                break;
            }
            case SchemeType::CKKS: {
                rns_tool.divide_and_round_q_last_ntt(encrypted.reference(), encrypted_size, destination.reference(), context_data->small_ntt_tables(), pool);
                break;
            }
            case SchemeType::BGV: {
                rns_tool.mod_t_and_divide_q_last_ntt(encrypted.reference(), encrypted_size, destination.reference(), context_data->small_ntt_tables(), pool);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::mod_switch_scale_to_next_internal] Scheme not implemented.");
            }
        }

        destination.is_ntt_form() = encrypted.is_ntt_form();
        if (scheme == SchemeType::CKKS) {
            // take the last modulus
            size_t id = parms.coeff_modulus().size() - 1;
            destination.scale() = encrypted.scale() / parms.coeff_modulus_host()[id].value();
        } else if (scheme == SchemeType::BGV) {
            destination.correction_factor() = utils::multiply_uint64_mod(
                encrypted.correction_factor(), rns_tool.inv_q_last_mod_t(), next_parms.plain_modulus_host()
            );
        }
    }

    __global__ static void kernel_mod_switch_drop_to(ConstSlice<uint64_t> source, size_t poly_count, size_t source_modulus_size, size_t remain_modulus_size, size_t degree, Slice<uint64_t> destination) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= remain_modulus_size * degree) return;
        size_t i = global_index % degree;
        size_t j = global_index / degree;
        for (size_t p = 0; p < poly_count; p++) {
            size_t source_index = (p * source_modulus_size + j) * degree + i;
            size_t dest_index = (p * remain_modulus_size + j) * degree + i;
            destination[dest_index] = source[source_index];
        }
    }

    void Evaluator::mod_switch_drop_to_internal(const Ciphertext& encrypted, Ciphertext& destination, ParmsID target_parms_id, MemoryPoolHandle pool) const {
        ParmsID parms_id = encrypted.parms_id();
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_scale_to_next_internal]", parms_id);
        const EncryptionParameters& parms = context_data->parms();
        SchemeType scheme = parms.scheme();
        if (scheme == SchemeType::CKKS) {
            check_is_ntt_form("[Evaluator::mod_switch_drop_to_internal]", encrypted);
        }
        if (!context_data->next_context_data().has_value()) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_next_internal] Next context data is not set.");
        }
        ContextDataPointer target_context_data = this->get_context_data("[Evaluator::mod_switch_drop_to_next_internal]", target_parms_id);
        const EncryptionParameters& target_parms = target_context_data->parms();
        if (!is_scale_within_bounds(encrypted.scale(), target_context_data)) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_internal] Scale out of bounds.");
        }
        
        size_t encrypted_size = encrypted.polynomial_count();
        size_t coeff_count = target_parms.poly_modulus_degree();
        size_t target_coeff_modulus_size = target_parms.coeff_modulus().size();

        destination = Ciphertext::like(encrypted, false, pool);

        destination.resize(this->context(), target_parms_id, encrypted_size, false, false);
        
        if (encrypted.on_device()) {
            size_t block_count = utils::ceil_div(target_coeff_modulus_size * coeff_count, utils::KERNEL_THREAD_COUNT);
            utils::set_device(encrypted.data().device_index());
            kernel_mod_switch_drop_to<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                encrypted.data().const_reference(), encrypted_size, parms.coeff_modulus().size(), target_coeff_modulus_size, coeff_count, destination.data().reference()
            );
            utils::stream_sync();
        } else {
            for (size_t p = 0; p < encrypted_size; p++) {
                for (size_t i = 0; i < coeff_count; i++) {
                    for (size_t j = 0; j < target_coeff_modulus_size; j++) {
                        size_t source_index = (p * parms.coeff_modulus().size() + j) * coeff_count + i;
                        size_t dest_index = (p * target_parms.coeff_modulus().size() + j) * coeff_count + i;
                        destination.data()[dest_index] = encrypted.data()[source_index];
                    }
                }
            }
        }

        destination.is_ntt_form() = encrypted.is_ntt_form();
        destination.scale() = encrypted.scale();
        destination.correction_factor() = encrypted.correction_factor();
    }

    void Evaluator::mod_switch_drop_to_plain_internal(const Plaintext& plain, Plaintext& destination, ParmsID target_parms_id, MemoryPoolHandle pool) const {
        if (!plain.is_ntt_form()) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_plain_internal] Plaintext is not in NTT form.");
        }
        ParmsID parms_id = plain.parms_id();
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_drop_to_plain_internal]", parms_id);
        
        if (!context_data->next_context_data().has_value()) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_plain_internal] Next context data is not set.");
        }
        ContextDataPointer target_context_data = this->get_context_data("[Evaluator::mod_switch_drop_to_plain_internal]", target_parms_id);
        const EncryptionParameters& target_parms = target_context_data->parms();
        if (!is_scale_within_bounds(plain.scale(), target_context_data)) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_plain_internal] Scale out of bounds.");
        }

        destination = Plaintext::like(plain, false, pool);
        destination.resize_rns(*context(), target_parms_id, false);

        if (plain.on_device()) {
            size_t block_count = utils::ceil_div(target_parms.coeff_modulus().size() * target_parms.poly_modulus_degree(), utils::KERNEL_THREAD_COUNT);
            utils::set_device(plain.data().device_index());
            kernel_mod_switch_drop_to<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                plain.data().const_reference(), 1, 
                context_data->parms().coeff_modulus().size(), 
                target_parms.coeff_modulus().size(), target_parms.poly_modulus_degree(), destination.data().reference()
            );
            utils::stream_sync();
        } else {
            for (size_t i = 0; i < target_parms.coeff_modulus().size(); i++) {
                for (size_t j = 0; j < target_parms.poly_modulus_degree(); j++) {
                    size_t source_index = i * context_data->parms().poly_modulus_degree() + j;
                    size_t dest_index = i * target_parms.poly_modulus_degree() + j;
                    destination.data()[dest_index] = plain.data()[source_index];
                }
            }
        }
    }

    void Evaluator::mod_switch_to_next(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::mod_switch_to_next]", encrypted);
        if (this->context()->last_parms_id() == encrypted.parms_id()) {
            throw std::invalid_argument("[Evaluator::mod_switch_to_next] End of modulus switching chain reached.");
        }
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: 
                this->mod_switch_scale_to_next_internal(encrypted, destination, pool);
                break;
            case SchemeType::CKKS: {
                auto context_data = this->get_context_data("[Evaluator::mod_switch_to_next]", encrypted.parms_id());
                if (!context_data->next_context_data().has_value()) {
                    throw std::invalid_argument("[Evaluator::mod_switch_to_next] Next context data is not set.");
                }
                auto target_context_data = context_data->next_context_data().value();
                this->mod_switch_drop_to_internal(encrypted, destination, target_context_data->parms_id(), pool);
                break;
            }
            case SchemeType::BGV:
                this->mod_switch_scale_to_next_internal(encrypted, destination, pool);
                break;
            default:
                throw std::logic_error("[Evaluator::mod_switch_to_next] Scheme not implemented.");
        }
    }

    void Evaluator::mod_switch_to(const Ciphertext& encrypted, const ParmsID& parms_id, Ciphertext& destination, MemoryPoolHandle pool) const {
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_to_inplace]", encrypted.parms_id());
        ContextDataPointer target_context_data = this->get_context_data("[Evaluator::mod_switch_to_inplace]", parms_id);
        if (context_data->chain_index() < target_context_data->chain_index()) {
            throw std::invalid_argument("[Evaluator::mod_switch_to_inplace] Cannot switch to a higher level.");
        }
        if (encrypted.parms_id() == parms_id) {
            destination = encrypted.clone(pool); return;
        }
        if (context_data->parms().scheme() == SchemeType::CKKS) {
            this->mod_switch_drop_to_internal(encrypted, destination, parms_id, pool);
        } else {
            bool first = true;
            while (true) {
                if (first) {this->mod_switch_to_next(encrypted, destination, pool); first = false;}
                else this->mod_switch_to_next_inplace(destination, pool);
                if (destination.parms_id() == parms_id) break;
            }
        }
    }

    void Evaluator::mod_switch_plain_to(const Plaintext& plain, const ParmsID& parms_id, Plaintext& destination, MemoryPoolHandle pool) const {
        if (!plain.is_ntt_form()) {
            throw std::invalid_argument("[Evaluator::mod_switch_plain_to_inplace] Plaintext is not in NTT form.");
        }
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_plain_to_inplace]", plain.parms_id());
        ContextDataPointer target_context_data = this->get_context_data("[Evaluator::mod_switch_plain_to_inplace]", parms_id);
        if (context_data->chain_index() < target_context_data->chain_index()) {
            throw std::invalid_argument("[Evaluator::mod_switch_plain_to_inplace] Cannot switch to a higher level.");
        }
        if (plain.parms_id() == parms_id) {
            destination = plain.clone(); return;
        }
        this->mod_switch_drop_to_plain_internal(plain, destination, parms_id, pool);
    }

    void Evaluator::rescale_to_next(const Ciphertext& encrypted, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::rescale_to_next]", encrypted);
        if (this->context()->last_parms_id() == encrypted.parms_id()) {
            throw std::invalid_argument("[Evaluator::rescale_to_next] End of modulus switching chain reached.");
        }
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: case SchemeType::BGV:
                throw std::invalid_argument("[Evaluator::rescale_to_next] Cannot rescale BFV/BGV ciphertext.");
                break;
            case SchemeType::CKKS:
                this->mod_switch_scale_to_next_internal(encrypted, destination, pool);
                break;
            default:
                throw std::logic_error("[Evaluator::rescale_to_next] Scheme not implemented.");
        }
    }
    
    void Evaluator::rescale_to(const Ciphertext& encrypted, const ParmsID& parms_id, Ciphertext& destination, MemoryPoolHandle pool) const {
        ContextDataPointer context_data = this->get_context_data("[Evaluator::rescale_to]", encrypted.parms_id());
        ContextDataPointer target_context_data = this->get_context_data("[Evaluator::rescale_to]", parms_id);
        if (context_data->chain_index() < target_context_data->chain_index()) {
            throw std::invalid_argument("[Evaluator::rescale_to] Cannot rescale to a higher level.");
        }
        while (encrypted.parms_id() != parms_id) {
            this->rescale_to_next(encrypted, destination, pool);
        }
    }

    
    void Evaluator::apply_galois(const Ciphertext& encrypted, size_t galois_element, const GaloisKeys& galois_keys, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::apply_galois_inplace]", encrypted);
        if (galois_keys.parms_id() != this->context()->key_parms_id()) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Galois keys has incorrect parms id.");
        }
        ContextDataPointer context_data = this->get_context_data("[Evaluator::apply_galois_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted_size = encrypted.polynomial_count();
        ContextDataPointer key_context_data = this->context()->key_context_data().value();
        const GaloisTool& galois_tool = key_context_data->galois_tool();

        if (!galois_keys.has_key(galois_element)) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Galois key not present.");
        }
        size_t m = coeff_count * 2;
        if ((galois_element & 1) == 0 || galois_element > m) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Galois element is not valid.");
        }
        if (encrypted_size > 2) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Ciphertext size must be 2.");
        }

        destination = Ciphertext::like(encrypted, false, pool);
        if (!encrypted.is_ntt_form()) {
            galois_tool.apply_ps(encrypted.const_polys(0, 1), 2, galois_element, coeff_modulus, destination.polys(0, 1));
        } else {
            galois_tool.apply_ntt_ps(encrypted.const_polys(0, 1), 2, coeff_modulus_size, galois_element, destination.polys(0, 1), pool);
        }

        this->switch_key_internal(encrypted, destination.poly(1), galois_keys.as_kswitch_keys(), GaloisKeys::get_index(galois_element), Evaluator::SwitchKeyDestinationAssignMethod::OverwriteExceptFirst, destination, pool);
    }
    
    void Evaluator::apply_galois_plain(const Plaintext& plain, size_t galois_element, Plaintext& destination, MemoryPoolHandle pool) const {
        ContextDataPointer context_data = plain.is_ntt_form()
            ? this->get_context_data("[Evaluator::apply_galois_plain_inplace]", plain.parms_id())
            : this->context()->key_context_data().value();
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        ContextDataPointer key_context_data = this->context()->key_context_data().value();
        const GaloisTool& galois_tool = key_context_data->galois_tool();
        
        size_t m = coeff_count * 2;
        if ((galois_element & 1) == 0 || galois_element > m) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Galois element is not valid.");
        }

        destination = Plaintext::like(plain, false, pool);
        if (!plain.is_ntt_form()) {
            if (context_data->is_ckks()) {
                galois_tool.apply_p(plain.const_poly(), galois_element, coeff_modulus, destination.reference());
            } else {
                galois_tool.apply(plain.const_poly(), galois_element, context_data->parms().plain_modulus(), destination.reference());
            }
        } else {
            galois_tool.apply_ntt_p(plain.const_poly(), coeff_modulus_size, galois_element, destination.reference(), pool);
        }
    }

    void Evaluator::rotate_internal(const Ciphertext& encrypted, int steps, const GaloisKeys& galois_keys, Ciphertext& destination, MemoryPoolHandle pool) const {
        ContextDataPointer context_data = this->get_context_data("[Evaluator::rotate_inplace_internal]", encrypted.parms_id());
        if (!context_data->qualifiers().using_batching) {
            throw std::invalid_argument("[Evaluator::rotate_inplace_internal] Batching must be enabled to use rotate.");
        }
        if (galois_keys.parms_id() != this->context()->key_parms_id()) {
            throw std::invalid_argument("[Evaluator::rotate_inplace_internal] Galois keys has incorrect parms id.");
        }
        if (steps == 0) return;
        const GaloisTool& galois_tool = context_data->galois_tool();
        if (galois_keys.has_key(galois_tool.get_element_from_step(steps))) {
            size_t element = galois_tool.get_element_from_step(steps);
            this->apply_galois(encrypted, element, galois_keys, destination, pool);
        } else {
            // Convert the steps to NAF: guarantees using smallest HW
            std::vector<int> naf_steps = utils::naf(steps);
            if (naf_steps.size() == 1) {
                throw std::invalid_argument("[Evaluator::rotate_inplace_internal] Galois key not present.");
            }
            bool done_flag = false;
            for (int naf_step : naf_steps) {
                if (!done_flag) {
                    this->rotate_internal(encrypted, naf_step, galois_keys, destination, pool);
                    done_flag = true;
                } else {
                    Ciphertext temp;
                    this->rotate_internal(destination, naf_step, galois_keys, temp, pool);
                    destination = std::move(temp);
                }
            }
        }
    }
    
    void Evaluator::conjugate_internal(const Ciphertext& encrypted, const GaloisKeys& galois_keys, Ciphertext& destination, MemoryPoolHandle pool) const {
        ContextDataPointer context_data = this->get_context_data("Evaluator::conjugate_inplace_internal", encrypted.parms_id());
        if (!context_data->qualifiers().using_batching) {
            throw std::logic_error("[Evaluator::conjugate_inplace_internal] Batching is not enabled.");
        }
        const GaloisTool& galois_tool = context_data->galois_tool();
        this->apply_galois(encrypted, galois_tool.get_element_from_step(0), galois_keys, destination, pool);
    }

    void Evaluator::negacyclic_shift(const Ciphertext& encrypted, size_t shift, Ciphertext& destination, MemoryPoolHandle pool) const {
        check_no_seed("[Evaluator::negacyclic_shift]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::negacyclic_shift]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();

        destination = Ciphertext::like(encrypted, false, pool);
        utils::negacyclic_shift_ps(
            encrypted.polys(0, encrypted.polynomial_count()),
            shift, encrypted.polynomial_count(), coeff_count, coeff_modulus, 
            destination.polys(0, destination.polynomial_count())
        );
    }
}