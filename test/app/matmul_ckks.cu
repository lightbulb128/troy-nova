#include <gtest/gtest.h>
#include <sstream>
#include "../test_adv.h"
#include "../../src/app/matmul.h"
#include "../test.h"

namespace matmul_ckks {

    using namespace troy;
    using namespace troy::linear;
    using tool::GeneralEncoder;
    using tool::GeneralVector;
    using tool::GeneralHeContext;
    using std::stringstream;
    using std::vector;

    void test_matmul(const GeneralHeContext& context, size_t m, size_t r, size_t n, bool pack_lwe, bool mod_switch_to_next) {
        SchemeType scheme = context.params_host().scheme();
        if (scheme != SchemeType::CKKS) {
            throw std::runtime_error("[test_matmul] Unsupported scheme");
        }
        double scale = context.scale();
        
        GeneralVector x = context.random_polynomial(m * r);
        GeneralVector w = context.random_polynomial(r * n);
        GeneralVector s = context.random_polynomial(m * n);
        MatmulHelper helper(m, r, n, context.params_host().poly_modulus_degree(), MatmulObjective::EncryptLeft, pack_lwe);

        HeContextPointer he = context.context();
        const CKKSEncoder& encoder = context.encoder().ckks();
        const Encryptor& encryptor = context.encryptor();
        const Evaluator& evaluator = context.evaluator();
        const Decryptor& decryptor = context.decryptor();
        GaloisKeys automorphism_key;
        if (pack_lwe) {
            automorphism_key = context.key_generator().create_automorphism_keys(false);
        }
        
        Cipher2d x_encrypted = helper.encrypt_inputs_doubles(encryptor, encoder, x.doubles().data(), std::nullopt, scale);
        Plain2d w_encoded = helper.encode_weights_doubles(encoder, w.doubles().data(), std::nullopt, scale, false);
        Plain2d s_encoded = helper.encode_outputs_doubles(encoder, s.doubles().data(), std::nullopt, scale * scale);

        stringstream x_serialized;
        x_encrypted.save(x_serialized, he);
        x_encrypted = Cipher2d::load_new(x_serialized, he);

        Cipher2d y_encrypted = helper.matmul(evaluator, x_encrypted, w_encoded);
        if (mod_switch_to_next) {
            y_encrypted.mod_switch_to_next_inplace(evaluator);
        }
        if (pack_lwe) {
            y_encrypted = helper.pack_outputs(evaluator, automorphism_key, y_encrypted);
        }

        y_encrypted.add_plain_inplace(evaluator, s_encoded);

        stringstream y_serialized;
        helper.serialize_outputs(evaluator, y_encrypted, y_serialized);
        y_encrypted = helper.deserialize_outputs(evaluator, y_serialized);

        vector<double> y_decrypted = helper.decrypt_outputs_doubles(encoder, decryptor, y_encrypted);   

        vector<double> y_truth(m * n, 0);
        for (size_t i = 0; i < m; i++) {
            for (size_t j = 0; j < n; j++) {
                for (size_t k = 0; k < r; k++) {
                    y_truth[i * n + j] += x.doubles()[i * r + k] * w.doubles()[k * n + j];
                }
                y_truth[i * n + j] += s.doubles()[i * n + j];
            }
        }

        GeneralVector decrypted(std::move(y_decrypted));
        GeneralVector truthv(std::move(y_truth));

        // std::cerr << "Truth:     " << truthv << std::endl;
        // std::cerr << "Decrypted: " << decrypted << std::endl;
        
        ASSERT_TRUE(truthv.near_equal(decrypted, context.tolerance()));
    }

    
    void test_matmul_reverse(const GeneralHeContext& context, size_t m, size_t r, size_t n, bool pack_lwe, bool mod_switch_to_next) {
        SchemeType scheme = context.params_host().scheme();
        if (scheme != SchemeType::CKKS) {
            throw std::runtime_error("[test_matmul] Unsupported scheme");
        }
        double scale = context.scale();
        
        GeneralVector x = context.random_polynomial(m * r);
        GeneralVector w = context.random_polynomial(r * n);
        GeneralVector s = context.random_polynomial(m * n);
        MatmulHelper helper(m, r, n, context.params_host().poly_modulus_degree(), MatmulObjective::EncryptRight, pack_lwe);

        HeContextPointer he = context.context();
        const CKKSEncoder& encoder = context.encoder().ckks();
        const Encryptor& encryptor = context.encryptor();
        const Evaluator& evaluator = context.evaluator();
        const Decryptor& decryptor = context.decryptor();
        GaloisKeys automorphism_key;
        if (pack_lwe) {
            automorphism_key = context.key_generator().create_automorphism_keys(false);
        }
        
        Plain2d x_encoded = helper.encode_inputs_doubles(encoder, x.doubles().data(), std::nullopt, scale, false);
        Cipher2d w_encrypted = helper.encrypt_weights_doubles(encryptor, encoder, w.doubles().data(), std::nullopt, scale);
        Plain2d s_encoded = helper.encode_outputs_doubles(encoder, s.doubles().data(), std::nullopt, scale * scale);

        stringstream w_serialized;
        w_encrypted.save(w_serialized, he);
        w_encrypted = Cipher2d::load_new(w_serialized, he);

        Cipher2d y_encrypted = helper.matmul_reverse(evaluator, x_encoded, w_encrypted);
        if (mod_switch_to_next) {
            y_encrypted.mod_switch_to_next_inplace(evaluator);
        }
        if (pack_lwe) {
            y_encrypted = helper.pack_outputs(evaluator, automorphism_key, y_encrypted);
        }

        y_encrypted.add_plain_inplace(evaluator, s_encoded);

        stringstream y_serialized;
        helper.serialize_outputs(evaluator, y_encrypted, y_serialized);
        y_encrypted = helper.deserialize_outputs(evaluator, y_serialized);

        vector<double> y_decrypted = helper.decrypt_outputs_doubles(encoder, decryptor, y_encrypted);   

        vector<double> y_truth(m * n, 0);
        for (size_t i = 0; i < m; i++) {
            for (size_t j = 0; j < n; j++) {
                for (size_t k = 0; k < r; k++) {
                    y_truth[i * n + j] += x.doubles()[i * r + k] * w.doubles()[k * n + j];
                }
                y_truth[i * n + j] += s.doubles()[i * n + j];
            }
        }

        GeneralVector decrypted(std::move(y_decrypted));
        GeneralVector truthv(std::move(y_truth));

        // std::cerr << "Truth:     " << truthv << std::endl;
        // std::cerr << "Decrypted: " << decrypted << std::endl;
        
        ASSERT_TRUE(truthv.near_equal(decrypted, context.tolerance()));
    }


    TEST(MatmulTest, HostCKKSMatmul) {
        GeneralHeContext ghe(false, SchemeType::CKKS, 1024, 0, { 60, 40, 40, 60 }, true, 0x123, 2, (double)(1<<20), 1e-2);
        srand(0);
        test_matmul(ghe, 4, 5, 6, false, false);
        test_matmul(ghe, 64, 128, 256, false, false);
        test_matmul(ghe, 4, 5, 6, true, false);
        test_matmul(ghe, 64, 128, 256, true, false);
    }

    TEST(MatmulTest, DeviceCKKSMatmul) {
        SKIP_WHEN_NO_CUDA_DEVICE;
        GeneralHeContext ghe(true, SchemeType::CKKS, 1024, 0, { 60, 40, 40, 60 }, true, 0x123, 2, (double)(1<<20), 1e-2);
        srand(0);
        test_matmul(ghe, 4, 5, 6, false, false);
        test_matmul(ghe, 64, 128, 256, false, false);
        // test_matmul(ghe, 400, 500, 600, false, false); // very slow!
        test_matmul(ghe, 4, 5, 6, true, false);
        test_matmul(ghe, 64, 128, 256, true, false);
        // test_matmul(ghe, 400, 500, 600, true, false); // very slow!
    }
    
    TEST(MatmulTest, HostCKKSMatmulReverse) {
        GeneralHeContext ghe(false, SchemeType::CKKS, 1024, 0, { 60, 40, 40, 60 }, true, 0x123, 2, (double)(1<<20), 1e-2);
        srand(0);
        test_matmul_reverse(ghe, 4, 5, 6, false, false);
        test_matmul_reverse(ghe, 64, 128, 256, false, false);
        test_matmul_reverse(ghe, 4, 5, 6, true, false);
        test_matmul_reverse(ghe, 64, 128, 256, true, false);
    }

    TEST(MatmulTest, DeviceCKKSMatmulReverse) {
        SKIP_WHEN_NO_CUDA_DEVICE;
        GeneralHeContext ghe(true, SchemeType::CKKS, 1024, 0, { 60, 40, 40, 60 }, true, 0x123, 2, (double)(1<<20), 1e-2);
        srand(0);
        test_matmul_reverse(ghe, 4, 5, 6, false, false);
        test_matmul_reverse(ghe, 64, 128, 256, false, false);
        // test_matmul(ghe, 400, 500, 600, false, false); // very slow!
        test_matmul_reverse(ghe, 4, 5, 6, true, false);
        test_matmul_reverse(ghe, 64, 128, 256, true, false);
        // test_matmul(ghe, 400, 500, 600, true, false); // very slow!
    }

}